#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "../../common/para.h"

// Num. of Channel
#define N_ch (TK_NUM * BT_NUM)
// Num. of sample
#define N_samp 8
#define N_col 64

double my_timer()
{
struct timeval time;
double _ret_val_0;
gettimeofday(( & time), 0);
_ret_val_0=(time.tv_sec+(time.tv_usec/1000000.0));
return _ret_val_0;
}

__global__ void FBCore(float *r, float *H, float *Vect_H, float *Vect_Dn, 
			float *Vect_Up, float *Vect_F, float *F, int threads, int size);
void FBComb(float *y, float *Vect_F, int *num_size, int index);
void h_FBCore(float *r, float *H, float *Vect_H, float *Vect_Dn, float *Vect_Up, float *Vect_F, float *F, int N_sim);

int main(){

  	float **r;
  	float **r_dev;
  	float **H;
  	float **H_dev;
  	float **F;
  	float **F_dev;

  	float **Vect_H; // output of the F
  	float **Vect_H_dev;
  	float **Vect_Dn; // output of the down sampler
  	float **Vect_Dn_dev;
  	float **Vect_Up; // output of the up sampler
  	float **Vect_Up_dev;
  	float **Vect_F; // this is the output of the
  	float **Vect_F_dev;
  	int num_thread[N_ch];
  	int num_size[N_ch];

  	float **h_Vect_F;

  
  	FILE *f;
	hipSetDevice(0);
  	setenv("CUDA_DEVICE_MAX_CONNECTIONS", "32", 1);

  	hipStream_t *filter_stream;
  
  	int i, j;
  	double start_timer, end_timer;
  
  	filter_stream = (hipStream_t*)malloc(N_ch*sizeof(hipStream_t));
  	for(i = 0; i < N_ch; i++){
    		checkCudaErrors(hipStreamCreate(&filter_stream[i]));
  	}

  	f = fopen("rand.txt", "r");
  	for(i = 0; i < N_ch; i++)
    		fscanf(f, "%1d", &num_thread[i]);

  	fclose(f);

  	for(i = 0; i < N_ch; i++)
    		num_size[i] = (num_thread[i]*16)*(num_thread[i]*16);


  	r = (float**)malloc(N_ch*sizeof(float*));
  	H = (float**)malloc(N_ch*sizeof(float*));
  	F = (float**)malloc(N_ch*sizeof(float*));
  	Vect_H = (float**)malloc(N_ch*sizeof(float*));
  	Vect_Dn = (float**)malloc(N_ch*sizeof(float*));
  	Vect_Up = (float**)malloc(N_ch*sizeof(float*));
  	Vect_F = (float**)malloc(N_ch*sizeof(float*));
  	r_dev = (float**)malloc(N_ch*sizeof(float*));
  	H_dev = (float**)malloc(N_ch*sizeof(float*));
  	F_dev = (float**)malloc(N_ch*sizeof(float*));
  	Vect_H_dev = (float**)malloc(N_ch*sizeof(float*));
  	Vect_Dn_dev = (float**)malloc(N_ch*sizeof(float*));
  	Vect_Up_dev = (float**)malloc(N_ch*sizeof(float*));
  	Vect_F_dev = (float**)malloc(N_ch*sizeof(float*));

  	Vect_F_dev = (float**)malloc(N_ch*sizeof(float*));
  	h_Vect_F = (float**)malloc(N_ch*sizeof(float*));



  	/*Memory allocation*/
  	for(i = 0; i < N_ch; i++){
    		checkCudaErrors(hipHostAlloc(&r[i], num_size[i]*sizeof(float), hipHostMallocDefault));
    		checkCudaErrors(hipMalloc(&r_dev[i], num_size[i]*sizeof(float)));
    		checkCudaErrors(hipHostAlloc(&H[i], N_col*sizeof(float), hipHostMallocDefault));
    		checkCudaErrors(hipMalloc(&H_dev[i], N_col*sizeof(float)));
    		checkCudaErrors(hipHostAlloc(&F[i], N_col*sizeof(float), hipHostMallocDefault));
    		checkCudaErrors(hipMalloc(&F_dev[i], N_col*sizeof(float)));

    		checkCudaErrors(hipHostAlloc(&Vect_H[i], num_size[i]*sizeof(float), hipHostMallocDefault));
    		checkCudaErrors(hipMalloc(&Vect_H_dev[i], num_size[i]*sizeof(float)));
    		checkCudaErrors(hipHostAlloc(&Vect_Dn[i], (num_size[i]/N_samp)*sizeof(float), hipHostMallocDefault));
    		checkCudaErrors(hipMalloc(&Vect_Dn_dev[i], (num_size[i]/N_samp)*sizeof(float)));
    		checkCudaErrors(hipHostAlloc(&Vect_Up[i], num_size[i]*sizeof(float), hipHostMallocDefault));
    		checkCudaErrors(hipMalloc(&Vect_Up_dev[i], num_size[i]*sizeof(float)));
    		checkCudaErrors(hipHostAlloc(&Vect_F[i], num_size[i]*sizeof(float), hipHostMallocDefault));
    		checkCudaErrors(hipMalloc(&Vect_F_dev[i], num_size[i]*sizeof(float)));

    		h_Vect_F[i] = (float*)malloc(num_size[i] * sizeof(float));

  	}

	printf("Filterbank inputs are generating\n");
  	/*init data*/
  	for(i = 0; i < N_ch; i++)
    		for(j = 0; j < num_size[i]; j++){
      			r[i][j] = j + 0.0001;
      			Vect_Up[i][j] = 0;
      			Vect_F[i][j] = 0;
      			Vect_H[i][j]=0;
      			h_Vect_F[i][j] = 0;
    		}

  	for(i = 0; i < N_ch; i++)
    		for(j = 0; j < N_col; j++){
      			H[i][j] = 0.0001;
      			F[i][j] = 0.0001;
    		}

  	// Data transfer to device
  	for(i = 0; i < N_ch; i++){
    		checkCudaErrors(hipMemcpyAsync(r_dev[i], r[i], num_size[i]*sizeof(float), hipMemcpyHostToDevice, filter_stream[i]));
    		checkCudaErrors(hipMemcpyAsync(Vect_Up_dev[i], Vect_Up[i], num_size[i]*sizeof(float), hipMemcpyHostToDevice, filter_stream[i]));
    		checkCudaErrors(hipMemcpyAsync(Vect_F_dev[i], Vect_F[i], num_size[i]*sizeof(float), hipMemcpyHostToDevice, filter_stream[i]));
    		checkCudaErrors(hipMemcpyAsync(Vect_H_dev[i], Vect_H[i], num_size[i]*sizeof(float), hipMemcpyHostToDevice, filter_stream[i]));
    		checkCudaErrors(hipMemcpyAsync(H_dev[i], H[i], N_col*sizeof(float), hipMemcpyHostToDevice, filter_stream[i]));
    		checkCudaErrors(hipMemcpyAsync(F_dev[i], F[i], N_col*sizeof(float), hipMemcpyHostToDevice, filter_stream[i]));
  	}
  	checkCudaErrors(hipDeviceSynchronize());

	printf("Filterbank CUDA baseline is running\n");
  	// task launch
  	start_timer = my_timer();
  	for(i = 0; i < N_ch; i++){
    		FBCore<<<1, num_thread[i]*32, 0, filter_stream[i]>>>(r_dev[i], H_dev[i], Vect_H_dev[i], 
			Vect_Dn_dev[i], Vect_Up_dev[i], Vect_F_dev[i], F_dev[i], num_thread[i]*32, num_size[i]);
  	}
  	checkCudaErrors(hipDeviceSynchronize());

  	end_timer = my_timer();
  	printf("Filterbank CUDA baseline Elapsed Time: %f Sec.\n", end_timer - start_timer);

  	// Data transfer back to host
  	for(i = 0; i < N_ch; i++){
    		checkCudaErrors(hipMemcpyAsync(Vect_F[i], Vect_F_dev[i], num_size[i]*sizeof(float), hipMemcpyDeviceToHost, filter_stream[i]));
  	}
  	checkCudaErrors(hipDeviceSynchronize());

	/*CPU tasks*/
	printf("CPU program running\n");
        start_timer = my_timer();
        for(i = 0; i < N_ch; i++){
                h_FBCore(r[i], H[i], Vect_H[i], Vect_Dn[i], Vect_Up[i], h_Vect_F[i], F[i], num_size[i]);
        }

        end_timer = my_timer();
        //printf("CPU Elapsed time:%f Sec.\n", end_timer - start_timer);

	/*Verify*/
        printf("Verify\n");
        int flag = 0;
        for(i = 0; i < N_ch; i++){
                for(j = 0; j < num_size[i]; j++){
                        if(abs(h_Vect_F[i][j] -  Vect_F[i][j]) > 1e-3){
                                printf("Error:%f, %f, %d\n", h_Vect_F[i][j], Vect_F[i][j], i);
                                flag = 1;
                                break;
                        }
                }
        }

        if(!flag) printf("Verify successfully\n");

  	/*Free Memory*/

  	for(i = 0; i < N_ch; i++){
    		checkCudaErrors(hipStreamDestroy(filter_stream[i]));
  	}

  	for(i = 0; i < N_ch; i++){ 
    		checkCudaErrors(hipHostFree(r[i]));
    		checkCudaErrors(hipFree(r_dev[i]));
    		checkCudaErrors(hipHostFree(H[i]));
    		checkCudaErrors(hipFree(H_dev[i]));
    		checkCudaErrors(hipHostFree(F[i]));
    		checkCudaErrors(hipFree(F_dev[i]));

    		checkCudaErrors(hipHostFree(Vect_H[i]));
    		checkCudaErrors(hipFree(Vect_H_dev[i]));
    		checkCudaErrors(hipHostFree(Vect_Dn[i]));
    		checkCudaErrors(hipFree(Vect_Dn_dev[i]));
   	 	checkCudaErrors(hipHostFree(Vect_Up[i]));
    		checkCudaErrors(hipFree(Vect_Up_dev[i]));
    		checkCudaErrors(hipHostFree(Vect_F[i]));
		checkCudaErrors(hipFree(Vect_F_dev[i]));

    		free(h_Vect_F[i]);
  	}

  	free(r);
  	free(H);
  	free(F);
  	free(Vect_H);
  	free(Vect_Dn);
  	free(Vect_Up);
  	free(Vect_F);
  	free(r_dev);
  	free(H_dev);
  	free(F_dev);
  	free(Vect_H_dev);
  	free(Vect_Dn_dev);
  	free(Vect_Up_dev);
  	free(Vect_F_dev);
	free(h_Vect_F);

  return 0;
}

void h_FBCore(float *r, float *H, float *Vect_H, float *Vect_Dn, float *Vect_Up, float *Vect_F, float *F, int N_sim){
        int j, k, p;

        //convolving H
        for (j=0; j< N_sim; j++)
        {
                for(k = 0; k < N_col; k++){
                        if((j-k)>=0){
                                Vect_H[j] += (r[j-k]*H[k]);
                        }
                }
        }

        //Down Sampling
        for (j=0; j < N_sim/N_samp; j++)
                Vect_Dn[j]=Vect_H[j*N_samp];

        //Up Sampling
        for (j=0; j < N_sim/N_samp;j++)
                Vect_Up[j*N_samp]=Vect_Dn[j];

        //convolving F
        for (j=0; j< N_sim; j++)
        {
                for(k = 0; k < N_col; k++){
                        if((j-k)>=0){
                                Vect_F[j]+=(F[k]*Vect_Up[j-k]);
                        }
                }
        }

}


__global__ void FBCore(float *r, float *H, float *Vect_H, float *Vect_Dn,
                        float *Vect_Up, float *Vect_F, float *F, int threads, int size){
  	int tid = threadIdx.x + blockIdx.x * blockDim.x;
  	int j, k;

  	//convolving H
  	if(tid < threads){
    		for (j=0; j< (size/threads); j++){
      			for(k = 0; k < N_col; k++){
        			if(((j*threads+tid)-k)>=0){
          				Vect_H[j*threads+tid] += (r[(j*threads+tid)-k]*H[k]);
        			}
      			}
    		}
  	}
  	__syncthreads();

  	//Down Sampling
  	if(tid < threads)
    		for (j=0; j < size/N_samp/threads; j++)
      			Vect_Dn[(j*threads+tid)]=Vect_H[(j*threads+tid)*N_samp];

  	//Up Sampling
  	if(tid < threads)
    		for (j=0; j < size/N_samp/threads;j++)
      			Vect_Up[(j*threads+tid)*N_samp]=Vect_Dn[(j*threads+tid)];
  	__syncthreads();

  	//convolving F
  	if(tid < threads){
    		for (j=0; j< (size/threads); j++){
      			for(k = 0; k < N_col; k++){
        			if(((j*threads+tid)-k)>=0){
          				Vect_F[j*threads+tid]+=(F[k]*Vect_Up[(j*threads+tid)-k]);
        			}
      			}
    		}
  	}
}
