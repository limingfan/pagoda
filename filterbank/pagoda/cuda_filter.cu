#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "runtime.cuh"
#include "filter.h"

double my_timer()
{
struct timeval time;
double _ret_val_0;
gettimeofday(( & time), 0);
_ret_val_0=(time.tv_sec+(time.tv_usec/1000000.0));
return _ret_val_0;
}

void h_FBCore(float *r, float *H, float *Vect_H, float *Vect_Dn, float *Vect_Up, float *Vect_F, float *F, int N_sim);

void FBComb(float *y, float *Vect_F, int N_sim){
  int j;
  //adding the results to the y matrix
    for (j=0; j < N_sim; j++)
      y[j]+=Vect_F[j];

}
int main(int argc, char *argv[]){

	float **r;
  	float **r_dev;
  	float *y;
  	float **H;
  	float **H_dev;
  	float **F;
 	float **F_dev;

  	float **Vect_H; // output of the F
  	float **Vect_H_dev;
  	float **Vect_Dn; // output of the down sampler
  	float **Vect_Dn_dev;
  	float **Vect_Up; // output of the up sampler
  	float **Vect_Up_dev;
  	float **Vect_F; // this is the output of the
  	float **Vect_F_dev;

  	float **h_Vect_F;
  	float *h_y;

	if(argc < 4){
		printf("Error input: filter length #channel #thread\n");
		exit(1);
	}

	int N_sim = atoi(argv[1]);
   	int N_ch = atoi(argv[2]);
	int TDD_NUM = atoi(argv[3]);
  
  	setenv("CUDA_DEVICE_MAX_CONNECTIONS", "32", 1);
  
  	int i, j;
  	double start_timer, end_timer;

  	runtime_init();
 
  	printf("Pagoda FilterBank: #task:%d, size:%d, #threads:%d\n", N_ch, N_sim, TDD_NUM); 

  	r = (float**)malloc(N_ch*sizeof(float*));
  	H = (float**)malloc(N_ch*sizeof(float*));
  	F = (float**)malloc(N_ch*sizeof(float*));
  	Vect_H = (float**)malloc(N_ch*sizeof(float*));
  	Vect_Dn = (float**)malloc(N_ch*sizeof(float*));
  	Vect_Up = (float**)malloc(N_ch*sizeof(float*));
  	Vect_F = (float**)malloc(N_ch*sizeof(float*));
  	r_dev = (float**)malloc(N_ch*sizeof(float*));
 	 H_dev = (float**)malloc(N_ch*sizeof(float*));
  	F_dev = (float**)malloc(N_ch*sizeof(float*));
  	Vect_H_dev = (float**)malloc(N_ch*sizeof(float*));
  	Vect_Dn_dev = (float**)malloc(N_ch*sizeof(float*));
  	Vect_Up_dev = (float**)malloc(N_ch*sizeof(float*));
  	Vect_F_dev = (float**)malloc(N_ch*sizeof(float*));
  	h_Vect_F = (float**)malloc(N_ch*sizeof(float*));
  


  	/*Memory allocation*/
  	for(i = 0; i < N_ch; i++){
    		checkCudaErrors(hipHostAlloc(&r[i], N_sim*sizeof(float), hipHostMallocDefault));
    		checkCudaErrors(hipMalloc(&r_dev[i], N_sim*sizeof(float)));
    		checkCudaErrors(hipHostAlloc(&H[i], N_col*sizeof(float), hipHostMallocDefault));
    		checkCudaErrors(hipMalloc(&H_dev[i], N_col*sizeof(float)));
    		checkCudaErrors(hipHostAlloc(&F[i], N_col*sizeof(float), hipHostMallocDefault));
    		checkCudaErrors(hipMalloc(&F_dev[i], N_col*sizeof(float)));

    		checkCudaErrors(hipHostAlloc(&Vect_H[i], N_sim*sizeof(float), hipHostMallocDefault));
    		checkCudaErrors(hipMalloc(&Vect_H_dev[i], N_sim*sizeof(float)));
    		checkCudaErrors(hipHostAlloc(&Vect_Dn[i], (N_sim/N_samp)*sizeof(float), hipHostMallocDefault));
    		checkCudaErrors(hipMalloc(&Vect_Dn_dev[i], (N_sim/N_samp)*sizeof(float)));
    		checkCudaErrors(hipHostAlloc(&Vect_Up[i], N_sim*sizeof(float), hipHostMallocDefault));
    		checkCudaErrors(hipMalloc(&Vect_Up_dev[i], N_sim*sizeof(float)));
    		checkCudaErrors(hipHostAlloc(&Vect_F[i], N_sim*sizeof(float), hipHostMallocDefault));
    		checkCudaErrors(hipMalloc(&Vect_F_dev[i], N_sim*sizeof(float)));
    		h_Vect_F[i] = (float*)malloc(N_sim * sizeof(float));
  	}

  	y = (float*)malloc(N_sim*sizeof(float));
  	h_y = (float*)malloc(N_sim*sizeof(float));

	//srand(time(NULL));
  	/*init data*/
  	for(i = 0; i < N_ch; i++)
    		for(j = 0; j < N_sim; j++){
      			r[i][j] = j + 0.0001;
      			y[j] = 0;
      			Vect_Up[i][j] = 0;
      			Vect_F[i][j] = 0;
      			Vect_H[i][j]=0;
      			h_Vect_F[i][j] = 0;
    		}

  	for(i = 0; i < N_ch; i++)
    		for(j = 0; j < N_col; j++){
      			H[i][j] = 0.0001;
      			F[i][j] = 0.0001;
    	}

  	double timer = 0.0;
  	start_timer = my_timer();

  	// Data transfer to device
  	for(i = 0; i < N_ch; i++){
    		checkCudaErrors(hipMemcpyAsync(r_dev[i], r[i], N_sim*sizeof(float), hipMemcpyHostToDevice, runtime_stream));
    		checkCudaErrors(hipMemcpyAsync(Vect_Up_dev[i], Vect_Up[i], N_sim*sizeof(float), hipMemcpyHostToDevice, runtime_stream));
    		checkCudaErrors(hipMemcpyAsync(Vect_F_dev[i], Vect_F[i], N_sim*sizeof(float), hipMemcpyHostToDevice, runtime_stream));
    		checkCudaErrors(hipMemcpyAsync(Vect_H_dev[i], Vect_H[i], N_sim*sizeof(float), hipMemcpyHostToDevice, runtime_stream));
    		checkCudaErrors(hipMemcpyAsync(H_dev[i], H[i], N_col*sizeof(float), hipMemcpyHostToDevice, runtime_stream));
    		checkCudaErrors(hipMemcpyAsync(F_dev[i], F[i], N_col*sizeof(float), hipMemcpyHostToDevice, runtime_stream));
  	}

  	checkCudaErrors(hipStreamSynchronize(runtime_stream));
  	end_timer = my_timer();
  	timer = (end_timer - start_timer);

  	// task launch
  	start_timer = my_timer();

  	for(i = 0; i < N_ch; i++){
    		taskLaunch(14, INT, TDD_NUM, INT, 1, INT, 0, INT, 1, INT, 0, FLOAT, r_dev[i],
			FLOAT, H_dev[i], FLOAT, Vect_H_dev[i], FLOAT, Vect_Dn_dev[i], FLOAT, Vect_Up_dev[i],
			FLOAT, Vect_F_dev[i], FLOAT, F_dev[i], INT, N_sim, INT, TDD_NUM);
  	}
  	waitAll(N_ch);

  	end_timer = my_timer();
  	printf("The GPU Elapsed Time: %f Sec.\n", end_timer - start_timer);

  	start_timer = my_timer();

  	// Data transfer back to host
  	for(i = 0; i < N_ch; i++){
    		checkCudaErrors(hipMemcpyAsync(Vect_F[i], Vect_F_dev[i], N_sim*sizeof(float), hipMemcpyDeviceToHost, runtime_stream));
  	}
  	checkCudaErrors(hipStreamSynchronize(runtime_stream));

  	end_timer = my_timer();
  	timer += (end_timer - start_timer);
  	printf("The memory time:%f Sec.\n", timer);
  
  	runtime_destroy();
  	runtime_free();

  	/*Merge process*/
  	for(i = 0; i < N_ch; i++){
    		FBComb(y, Vect_F[i], N_sim);
  	}

  	/*CPU tasks*/
  	start_timer = my_timer();
  	for(i = 0; i < N_ch; i++){
    		h_FBCore(r[i], H[i], Vect_H[i], Vect_Dn[i], Vect_Up[i], h_Vect_F[i], F[i], N_sim);
  	}

  	end_timer = my_timer();
  	printf("CPU Elapsed time:%f Sec.\n", end_timer - start_timer);

  	/*Merge process*/
  	for(i = 0; i < N_ch; i++){
    		FBComb(h_y, h_Vect_F[i], N_sim);
  	}

  	/*Verify*/
  	printf("Verify\n");
	long long flag = 0;
  	for(i = 0; i < N_sim; i++){
    		if(abs(h_y[i] -  y[i]) > 1e-3){
			printf("Error:%f, %f, %d\n", h_y[i], y[i], i);
			break;
    		}
		flag ++;
  	}

	if(flag == N_sim) printf("Verify successfully\n");

  	/*Free Memory*/

  	for(i = 0; i < N_ch; i++){ 
    		checkCudaErrors(hipHostFree(r[i]));
    		checkCudaErrors(hipFree(r_dev[i]));
    		checkCudaErrors(hipHostFree(H[i]));
    		checkCudaErrors(hipFree(H_dev[i]));
    		checkCudaErrors(hipHostFree(F[i]));
    		checkCudaErrors(hipFree(F_dev[i]));

    		checkCudaErrors(hipHostFree(Vect_H[i]));
    		checkCudaErrors(hipFree(Vect_H_dev[i]));
    		checkCudaErrors(hipHostFree(Vect_Dn[i]));
    		checkCudaErrors(hipFree(Vect_Dn_dev[i]));
    		checkCudaErrors(hipHostFree(Vect_Up[i]));
    		checkCudaErrors(hipFree(Vect_Up_dev[i]));
    		checkCudaErrors(hipHostFree(Vect_F[i]));
    		free(h_Vect_F[i]);
  	}

  	free(y);
  	free(h_y);
  	free(r);
  	free(H);
  	free(F);
  	free(Vect_H);
  	free(Vect_Dn);
  	free(Vect_Up);
  	free(Vect_F);
  	free(r_dev);
  	free(H_dev);
  	free(F_dev);
  	free(Vect_H_dev);
  	free(Vect_Dn_dev);
  	free(Vect_Up_dev);
  	free(Vect_F_dev);
  	free(h_Vect_F);

  	return 0;
}

void h_FBCore(float *r, float *H, float *Vect_H, float *Vect_Dn, float *Vect_Up, float *Vect_F, float *F, int N_sim){
	int j, k, p;

  	//convolving H
  	for (j=0; j< N_sim; j++)
  	{
      		for(k = 0; k < N_col; k++){
        		if((j-k)>=0){
          			Vect_H[j] += (r[j-k]*H[k]);
        		}
      		}
  	}

  	//Down Sampling
  	for (j=0; j < N_sim/N_samp; j++)
    		Vect_Dn[j]=Vect_H[j*N_samp];

  	//Up Sampling
  	for (j=0; j < N_sim/N_samp;j++)
    		Vect_Up[j*N_samp]=Vect_Dn[j];

  	//convolving F
  	for (j=0; j< N_sim; j++)
  	{
      		for(k = 0; k < N_col; k++){
        		if((j-k)>=0){
          			Vect_F[j]+=(F[k]*Vect_Up[j-k]);
        		}
      		}
  	}

}
