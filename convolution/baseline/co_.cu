#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "con.h"

double my_timer()
{
struct timeval time;
double _ret_val_0;
gettimeofday(( & time), 0);
_ret_val_0=(time.tv_sec+(time.tv_usec/1000000.0));
return _ret_val_0;
}

__global__ void convolutionRowsGPU(float *h_Dst, float *h_Src, float *h_Kernel, int kernelR, int imageW, int imageH, int TD_NUM);
__global__ void convolutionColumnsGPU(float *h_Dst, float *h_Src, float *h_Kernel,int kernelR, int imageW, int imageH, int TD_NUM);

void convolutionRowCPU( float *h_Dst, float *h_Src, float *h_Kernel, int kernelR, int imageW, int imageH);
void convolutionColumnCPU( float *h_Dst, float *h_Src, float *h_Kernel, int kernelR, int imageW, int imageH);

int main(int argc, char *argv[]){

	float **h_Kernel, **h_Input, **d_Buffer, **h_OutputGPU, **h_Buffer, **h_OutputCPU;
	float **d_Output, **d_Kernel, **d_Input;

	int i, j;
	double start_timer, end_timer;

	if(argc < 5){
                printf("Input option: ./convolution imageW imageH #thread #task\n");
                exit(1);
        }
        int imageW = atoi(argv[1]);
        int imageH = atoi(argv[2]);
        int TD_NUM = atoi(argv[3]);
        int NUM_TASK = atoi(argv[4]);

	printf("CUDA baseline Conv: image=%d x %d, #thread:%d, #task:%d\n", imageW, imageH, TD_NUM, NUM_TASK);

	setenv("CUDA_DEVICE_MAX_CONNECTIONS", "32", 1);

	printf("Initializing data...\n");

	hipStream_t *con_stream;
	con_stream = (hipStream_t*)malloc(NUM_TASK*sizeof(con_stream));

	for(i = 0; i < NUM_TASK; i++){
    		checkCudaErrors(hipStreamCreate(&con_stream[i]));
  	}


	h_Kernel    = (float **)malloc(NUM_TASK * sizeof(float*));
	h_Input     = (float **)malloc(NUM_TASK * sizeof(float*));
	d_Buffer    = (float **)malloc(NUM_TASK * sizeof(float*));
	h_OutputGPU = (float **)malloc(NUM_TASK * sizeof(float*));
	h_OutputCPU = (float **)malloc(NUM_TASK * sizeof(float*));
	h_Buffer    = (float **)malloc(NUM_TASK * sizeof(float*));
	d_Output = (float **)malloc(NUM_TASK * sizeof(float*));
	d_Kernel = (float **)malloc(NUM_TASK * sizeof(float*));
	d_Input = (float **)malloc(NUM_TASK * sizeof(float*));

	for(i = 0; i < NUM_TASK; i++){
  		//h_Kernel[i]    = (float *)malloc(KERNEL_LENGTH * sizeof(float));
  		checkCudaErrors(hipHostAlloc(&h_Kernel[i], KERNEL_LENGTH*sizeof(float), hipHostMallocDefault));
  		//h_Input[i]     = (float *)malloc(imageW * imageH * sizeof(float));
  		checkCudaErrors(hipHostAlloc(&h_Input[i], imageW * imageH*sizeof(float), hipHostMallocDefault));
 		// h_Buffer[i]    = (float *)malloc(imageW * imageH * sizeof(float));
  		checkCudaErrors(hipMalloc(&d_Buffer[i], imageW * imageH * sizeof(float)));
  		//h_OutputGPU[i] = (float *)malloc(imageW * imageH * sizeof(float));
  		checkCudaErrors(hipHostAlloc(&h_OutputGPU[i], imageW * imageH*sizeof(float), hipHostMallocDefault));
  		checkCudaErrors(hipMalloc(&d_Output[i], imageW * imageH * sizeof(float)));
  		checkCudaErrors(hipMalloc(&d_Input[i], imageW * imageH * sizeof(float)));
  		checkCudaErrors(hipMalloc(&d_Kernel[i], KERNEL_LENGTH * sizeof(float)));

		h_Buffer[i] = (float*)malloc(imageW * imageH * sizeof(float));
  		h_OutputCPU[i] = (float*)malloc(imageW * imageH * sizeof(float));

	}

	for(i = 0; i < NUM_TASK;i++){
  		for (j = 0; j < KERNEL_LENGTH; j++){
    			h_Kernel[i][j] = (float)j/KERNEL_LENGTH;
  		}
	}

	for(i = 0; i < NUM_TASK;i++){
  		for (j = 0; j < imageW * imageH; j++){
    			//h_Input[i][j] = (float)((j/imageW)%2);
			h_Input[i][j] = rand()%100;

  		}
	}

	//mem. copy
	for(i = 0; i < NUM_TASK; i++){
  		checkCudaErrors(hipMemcpyAsync(d_Kernel[i], h_Kernel[i], KERNEL_LENGTH*sizeof(float), hipMemcpyHostToDevice, con_stream[i%32]));
  		checkCudaErrors(hipMemcpyAsync(d_Input[i], h_Input[i], imageW * imageH*sizeof(float), hipMemcpyHostToDevice, con_stream[i%32]));

	}
	checkCudaErrors(hipDeviceSynchronize());


	start_timer = my_timer();
	for(i = 0; i < NUM_TASK; i++){
  		convolutionRowsGPU<<<1, TD_NUM, 0, con_stream[i%32]>>>(d_Buffer[i], d_Input[i], d_Kernel[i], KERNEL_RADIUS, imageW, imageH, TD_NUM);
	}
	checkCudaErrors(hipDeviceSynchronize());
	for(i = 0; i < NUM_TASK; i++){
  		convolutionColumnsGPU<<<1, TD_NUM, 0, con_stream[i%32]>>>(d_Output[i], d_Buffer[i], d_Kernel[i],KERNEL_RADIUS, imageW, imageH, TD_NUM);
	}
	checkCudaErrors(hipDeviceSynchronize());
	end_timer = my_timer();
	printf("GPU elapsed time:%lf Sec.\n", end_timer - start_timer);

	start_timer = my_timer();
	for(i = 0; i < NUM_TASK; i++){
  		checkCudaErrors(hipMemcpyAsync(h_OutputGPU[i], d_Output[i], imageW * imageH*sizeof(float), hipMemcpyDeviceToHost, con_stream[i%32]));

	}
	checkCudaErrors(hipDeviceSynchronize());

	printf("CPU convolution Start\n");
  	start_timer = my_timer();

  	for(i = 0; i < NUM_TASK; i++){
    		convolutionRowCPU( h_Buffer[i], h_Input[i], h_Kernel[i], KERNEL_RADIUS, imageW, imageH);
  	}
  	for(i = 0; i < NUM_TASK; i++){
    		convolutionColumnCPU( h_OutputCPU[i], h_Buffer[i], h_Kernel[i], KERNEL_RADIUS, imageW, imageH);
  	}
  	end_timer = my_timer();
  	printf("CPU exec.time:%lf Sec.\n", end_timer - start_timer);

	//verification
  	printf("verify\n");
	long long flag = 0;
  	for(i = 0; i < NUM_TASK; i++){
    		for(j = 0; j < imageW * imageH; j++){
        		if(fabs(h_OutputCPU[i][j] - h_OutputGPU[i][j] > 0.1)){
          			printf("Error:%f, %f, %d, %d\n", h_OutputCPU[i][j], h_OutputGPU[i][j]);
          		break;
        		}
			flag ++;
    		}
  	}
	if(flag == (NUM_TASK * imageW * imageH)) printf("verify successfully\n");

	//free mem.
	for(i = 0; i < NUM_TASK; i++){
  		checkCudaErrors(hipStreamDestroy(con_stream[i]));
	}
	for(i = 0; i < NUM_TASK; i++){
  		//free(h_Buffer[i]);
  		checkCudaErrors(hipFree(d_Buffer[i]));
  		//free(h_Input[i]);
  		checkCudaErrors(hipHostFree(h_Input[i]));
  		//free(h_Kernel[i]);
  		checkCudaErrors(hipHostFree(h_Kernel[i]));
  		//free(h_OutputGPU[i]);
  		checkCudaErrors(hipHostFree(h_OutputGPU[i]));
  		checkCudaErrors(hipFree(d_Kernel[i]));
  		checkCudaErrors(hipFree(d_Output[i]));
  		checkCudaErrors(hipFree(d_Input[i]));

		free(h_Buffer[i]);
  		free(h_OutputCPU[i]);

	}

	free(d_Buffer);
	free(h_Input);
	free(h_Kernel);
	free(d_Kernel);
	free(d_Output);
	free(d_Input);
	free(h_OutputGPU);
	free(h_Buffer);
	free(h_OutputCPU);

return 0;
}

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionRowsGPU(
    float *h_Dst,
    float *h_Src,
    float *h_Kernel,
    int kernelR,
    int imageW,
    int imageH,
    int TDD_NUM
)
{
    int y = blockIdx.x * blockDim.x + threadIdx.x;

    if(y < TDD_NUM){
        for (int x = 0; x < (imageW*imageH)/TDD_NUM; x++)
        {
            float sum = 0;

            for (int k = -kernelR; k <= kernelR; k++)
            {
                int d = ((x*TDD_NUM+y)%imageW) + k;

                if (d >= 0 && d < imageW)
                    sum += h_Src[((x*TDD_NUM+y)/imageH) * imageW + d] * h_Kernel[kernelR - k];
            }

            h_Dst[((x*TDD_NUM+y)/imageH) * imageW + ((x*TDD_NUM+y)%imageW)] = sum;
        }
    }

}

////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionColumnsGPU(
    float *h_Dst,
    float *h_Src,
    float *h_Kernel,
    int kernelR,
    int imageW,
    int imageH,
    int TDD_NUM
)
{
    int y = blockIdx.x * blockDim.x + threadIdx.x;

    if(y < TDD_NUM){
        for (int x = 0; x < (imageW * imageH)/TDD_NUM; x++)
        {
            float sum = 0;

            for (int k = -kernelR; k <= kernelR; k++)
            {
                int d = ((x*TDD_NUM+y)/imageH) + k;

                if (d >= 0 && d < imageH)
                    sum += h_Src[d * imageW + ((x*TDD_NUM+y)%imageW)] * h_Kernel[kernelR - k];
            }

            h_Dst[((x*TDD_NUM+y)/imageH) * imageW + ((x*TDD_NUM+y)%imageW)] = sum;
        }
    }

    
}

void convolutionRowCPU( float *h_Dst, float *h_Src, float *h_Kernel, int kernelR, int imageW, int imageH)
{
    for (int y = 0; y < imageH; y++)
        for (int x = 0; x < imageW; x++)
        {
            float sum = 0;

            for (int k = -kernelR; k <= kernelR; k++)
            {
                int d = x + k;

                if (d >= 0 && d < imageW)
                    sum += h_Src[y * imageW + d] * h_Kernel[kernelR - k];
            }

            h_Dst[y * imageW + x] = sum;
        }
}

void convolutionColumnCPU( float *h_Dst, float *h_Src, float *h_Kernel, int kernelR, int imageW, int imageH)
{
    for (int y = 0; y < imageH; y++)
        for (int x = 0; x < imageW; x++)
        {
            float sum = 0;

            for (int k = -kernelR; k <= kernelR; k++)
            {
                int d = y + k;

                if (d >= 0 && d < imageH)
                    sum += h_Src[d * imageW + x] * h_Kernel[kernelR - k];
            }

            h_Dst[y * imageW + x] = sum;
        }
}

