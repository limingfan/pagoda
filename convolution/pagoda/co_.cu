#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "con.h"
#include "runtime.cuh"

double my_timer()
{
struct timeval time;
double _ret_val_0;
gettimeofday(( & time), 0);
_ret_val_0=(time.tv_sec+(time.tv_usec/1000000.0));
return _ret_val_0;
}

void convolutionRowCPU( float *h_Dst, float *h_Src, float *h_Kernel, int kernelR, int imageW, int imageH);
void convolutionColumnCPU( float *h_Dst, float *h_Src, float *h_Kernel, int kernelR, int imageW, int imageH);

int main(int argc, char *argv[]){

	float **h_Kernel, **h_Input, **d_Buffer, **h_OutputGPU, **h_Buffer, **h_OutputCPU;
	float **d_Output, **d_Kernel, **d_Input;

	int i, j;
	double start_timer, end_timer;

	if(argc < 5){
		printf("Input option: ./convolution imageW imageH #thread #task\n");
		exit(1);
	}
	int imageW = atoi(argv[1]);
	int imageH = atoi(argv[2]);
	int TDD_NUM = atoi(argv[3]);
   	int NUM_TASK = atoi(argv[4]);

	printf("Pagoda Conv: image=%d x %d, #thread:%d, #task:%d\n", imageW, imageH, TDD_NUM, NUM_TASK);

	setenv("CUDA_DEVICE_MAX_CONNECTIONS", "32", 1);

	printf("Initializing data...\n");
	runtime_init();

	h_Kernel    = (float **)malloc(NUM_TASK * sizeof(float*));
	h_Input     = (float **)malloc(NUM_TASK * sizeof(float*));
	d_Buffer    = (float **)malloc(NUM_TASK * sizeof(float*));
	h_OutputGPU = (float **)malloc(NUM_TASK * sizeof(float*));
	h_OutputCPU = (float **)malloc(NUM_TASK * sizeof(float*));
	h_Buffer    = (float **)malloc(NUM_TASK * sizeof(float*));
	d_Output = (float **)malloc(NUM_TASK * sizeof(float*));
	d_Kernel = (float **)malloc(NUM_TASK * sizeof(float*));
	d_Input = (float **)malloc(NUM_TASK * sizeof(float*));

	for(i = 0; i < NUM_TASK; i++){
  		checkCudaErrors(hipHostAlloc(&h_Kernel[i], KERNEL_LENGTH*sizeof(float), hipHostMallocDefault));
  		checkCudaErrors(hipHostAlloc(&h_Input[i], imageW * imageH*sizeof(float), hipHostMallocDefault));
  		checkCudaErrors(hipMalloc(&d_Buffer[i], imageW * imageH * sizeof(float)));
  		checkCudaErrors(hipHostAlloc(&h_OutputGPU[i], imageW * imageH*sizeof(float), hipHostMallocDefault));
  		checkCudaErrors(hipMalloc(&d_Output[i], imageW * imageH * sizeof(float)));
  		checkCudaErrors(hipMalloc(&d_Input[i], imageW * imageH * sizeof(float)));
  		checkCudaErrors(hipMalloc(&d_Kernel[i], KERNEL_LENGTH * sizeof(float)));

  		h_Buffer[i] = (float*)malloc(imageW * imageH * sizeof(float));
  		h_OutputCPU[i] = (float*)malloc(imageW * imageH * sizeof(float));
	}

	for(i = 0; i < NUM_TASK;i++){
  		for (j = 0; j < KERNEL_LENGTH; j++){
    			h_Kernel[i][j] = (float)j/KERNEL_LENGTH;
  		}
	}

	for(i = 0; i < NUM_TASK;i++){
  		for (j = 0; j < imageW * imageH; j++){
    			h_Input[i][j] = rand()%100;

  		}
	}

	//mem. copy

	for(i = 0; i < NUM_TASK; i++){
  		checkCudaErrors(hipMemcpyAsync(d_Kernel[i], h_Kernel[i], KERNEL_LENGTH*sizeof(float), hipMemcpyHostToDevice, runtime_stream));
  		checkCudaErrors(hipMemcpyAsync(d_Input[i], h_Input[i], imageW * imageH*sizeof(float), hipMemcpyHostToDevice, runtime_stream));

	}
	checkCudaErrors(hipStreamSynchronize(runtime_stream));

	start_timer = my_timer();
	for(i = 0; i < NUM_TASK; i++){
  		taskLaunch(11, INT, TDD_NUM, INT, 1, INT, 0, INT, 0, INT, 0, FLOAT, d_Buffer[i], FLOAT, d_Input[i], FLOAT, d_Kernel[i], INT, imageW, INT, imageH, INT, TDD_NUM);
	}

	waitAll(NUM_TASK);
	for(i = 0; i < NUM_TASK; i++){
  		taskLaunch(11, INT, TDD_NUM, INT, 1, INT, 0, INT, 0, INT, 1, FLOAT, d_Output[i], FLOAT, d_Buffer[i], FLOAT, d_Kernel[i], INT, imageW, INT, imageH, INT, TDD_NUM);
	}	
	waitAll(NUM_TASK);

	end_timer = my_timer();
	printf("GPU elapsed time:%lf Sec.\n", end_timer - start_timer);

	for(i = 0; i < NUM_TASK; i++){
  		checkCudaErrors(hipMemcpyAsync(h_OutputGPU[i], d_Output[i], imageW * imageH*sizeof(float), hipMemcpyDeviceToHost, runtime_stream));
	}
	checkCudaErrors(hipStreamSynchronize(runtime_stream));


	runtime_destroy();
	runtime_free();

	printf("CPU convolution Start\n");
  	start_timer = my_timer();
  	for(i = 0; i < NUM_TASK; i++){
    		convolutionRowCPU( h_Buffer[i], h_Input[i], h_Kernel[i], KERNEL_RADIUS, imageW, imageH);
  	}

  	for(i = 0; i < NUM_TASK; i++){
    		convolutionColumnCPU( h_OutputCPU[i], h_Buffer[i], h_Kernel[i], KERNEL_RADIUS, imageW, imageH);
  	}
  	end_timer = my_timer();
  	printf("CPU exec.time:%lf Sec.\n", end_timer - start_timer);


	//verification
  	printf("verify\n");
	long long flag = 0;
  	for(i = 0; i < NUM_TASK; i++){
    		for(j = 0; j < imageW * imageH; j++){
        		if(fabs(h_OutputCPU[i][j] - h_OutputGPU[i][j] > 0.1)){
          			printf("Error:%f, %f, %d, %d\n", h_OutputCPU[i][j], h_OutputGPU[i][j]);
          			break;
        		}
			flag ++;
    		}
  	}
	if(flag == (NUM_TASK * imageW * imageH)) printf("verify successfully\n");

	//free mem.

	for(i = 0; i < NUM_TASK; i++){
  		checkCudaErrors(hipFree(d_Buffer[i]));
  		checkCudaErrors(hipHostFree(h_Input[i]));
  		checkCudaErrors(hipHostFree(h_OutputGPU[i]));
  		checkCudaErrors(hipFree(d_Kernel[i]));
  		checkCudaErrors(hipFree(d_Output[i]));
  		checkCudaErrors(hipFree(d_Input[i]));

  		free(h_Buffer[i]);
  		free(h_OutputCPU[i]);

	}

	free(d_Buffer);
	free(h_Input);
	free(h_Kernel);
	free(d_Kernel);
	free(d_Output);
	free(d_Input);
	free(h_OutputGPU);
	free(h_Buffer);
	free(h_OutputCPU);

	return 0;
}

void convolutionRowCPU( float *h_Dst, float *h_Src, float *h_Kernel, int kernelR, int imageW, int imageH)
{
    for (int y = 0; y < imageH; y++)
        for (int x = 0; x < imageW; x++)
        {
            float sum = 0;

            for (int k = -kernelR; k <= kernelR; k++)
            {
                int d = x + k;

                if (d >= 0 && d < imageW)
                    sum += h_Src[y * imageW + d] * h_Kernel[kernelR - k];
            }

            h_Dst[y * imageW + x] = sum;
        }
}

void convolutionColumnCPU( float *h_Dst, float *h_Src, float *h_Kernel, int kernelR, int imageW, int imageH)
{
    for (int y = 0; y < imageH; y++)
        for (int x = 0; x < imageW; x++)
        {
            float sum = 0;

            for (int k = -kernelR; k <= kernelR; k++)
            {
                int d = y + k;

                if (d >= 0 && d < imageH)
                    sum += h_Src[d * imageW + x] * h_Kernel[kernelR - k];
            }

            h_Dst[y * imageW + x] = sum;
        }
}

