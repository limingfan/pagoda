#include "hip/hip_runtime.h"
# include <stdlib.h>
# include <stdio.h>
# include <math.h>
# include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "runtime.cuh"
#include "mand.h"

double my_timer()
{
struct timeval time;
double _ret_val_0;
gettimeofday(( & time), 0);
_ret_val_0=(time.tv_sec+(time.tv_usec/1000000.0));
return _ret_val_0;
}

/*
  Determine the coloring of each pixel.
*/

void explodeCPU ( float x, float y, int *value){
        int k;
        float x1;
        float x2;
        float y1;
        float y2;
        *value = 0;

        x1 = x;
        y1 = y;

        for ( k = 1; k <= count_max; k++ )
        {
                x2 = x1 * x1 - y1 * y1 + x;
                y2 = 2.0 * x1 * y1 + y;

                if ( x2 < -2.0 || 2.0 < x2 || y2 < -2.0 || 2.0 < y2 )
                {
                        *value = k;
                        break;
                }
                x1 = x2;
                y1 = y2;
        }
}

void get_pixelCPU(int *count, float index){

        int i, j;
        float x, y;
        for ( i = 0; i < n; i++ )
        {
                for ( j = 0; j < n; j++ )
                {
                        x = ( ( float ) (     j     ) * (x_max + index)
                                + ( float ) ( n - j - 1 ) * (x_min + index) )
                                / ( float ) ( n     - 1 );

                        y = ( ( float ) (     i     ) * (y_max + index)
                                + ( float ) ( n - i - 1 ) * (y_min + index) )
                                / ( float ) ( n     - 1 );

                        explodeCPU ( x, y, &count[i + j * n] );
                }
        }

}
int det_pixel(int *c_max, int *count){

  int i, j;

  *c_max = 0;
  for ( j = 0; j < n; j++ )
  {
    for ( i = 0; i < n; i++ )
    {
      if ( *c_max < count[i+j*n] )
      {
        *c_max = count[i+j*n];
      }
    }
  }
}

/*
  Set the image data.
*/

void set_img(int *r, int *g, int *b, int *count, int c_max){
  int i, j;
  int c;

  for ( i = 0; i < n; i++ )
  {
    for ( j = 0; j < n; j++ )
    {
      if ( count[i+j*n] % 2 == 1 )
      {
        r[i+j*n] = 255;
        g[i+j*n] = 255;
        b[i+j*n] = 255;
      }
      else
      {
        c = ( int ) ( 255.0 * sqrt ( sqrt ( sqrt (
          ( ( double ) ( count[i+j*n] ) / ( double ) ( c_max ) ) ) ) ) );
        r[i+j*n] = 3 * c / 5;
        g[i+j*n] = 3 * c / 5;
        b[i+j*n] = c;
      }
    }
  }  
}

int main(int argc, char *argv[]){
  	int i, j, k;
  	int **r, **g, **b;
  	int *c_max;
  	int **count;
	int **countCPU;
  	int **count_dev;
  	float *task_indx;
  	float *task_indx_dev;

  	double start_timer, end_timer;

	if(argc < 3){
		printf("Error input options:./mand #thread #task");
		exit(1);
	}
	int TDD_NUM = atoi(argv[1]);
	int task = atoi(argv[2]);
	printf("Pagoda Mandelbrot:image size: %d x %d, #thread:%d, #task:%d\n", n, n, TDD_NUM, task);

  	count =  (int**)malloc(task * sizeof(int *));
	countCPU =  (int**)malloc(task * sizeof(int *));
  	count_dev = (int**)malloc(task * sizeof(int *));
  	r =  (int**)malloc(task * sizeof(int *));
  	g =  (int** )malloc(task * sizeof(int *));
  	b =  (int**)malloc(task * sizeof(int *));
  	c_max = (int*)malloc(task * sizeof(int));
  	task_indx = (float*)malloc(task * sizeof(float));
  	checkCudaErrors(hipMalloc(&task_indx_dev, task *sizeof(float)));
  	for(i = 0; i < task; i++) task_indx[i] = (float)(i/(task/2.0));

  	setenv("CUDA_DEVICE_MAX_CONNECTIONS", "32", 1);

  	runtime_init();

  	for(i = 0; i < task; i++){
		countCPU[i] = ( int * ) malloc ( n * n * sizeof ( int ) );
    		checkCudaErrors(hipHostAlloc(&count[i], n * n *sizeof(int), NULL));
    		checkCudaErrors(hipMalloc(&count_dev[i], n * n *sizeof(int)));
    		r[i] = ( int * ) malloc ( n * n * sizeof ( int ) );
    		g[i] = ( int * ) malloc ( n * n * sizeof ( int ) );
    		b[i] = ( int * ) malloc ( n * n * sizeof ( int ) );
  	}
  
  	checkCudaErrors(hipMemcpyAsync(task_indx_dev, task_indx, task*sizeof(float), hipMemcpyHostToDevice, runtime_stream));
  	checkCudaErrors(hipStreamSynchronize(runtime_stream));

  	start_timer = my_timer(); 
  	//Carry out the iteration for each pixel, determining COUNT.
  	for(i = 0 ; i < task ; i++){
    		taskLaunch(8, INT, TDD_NUM, INT, 1, INT, 0, INT, 0, INT, 0, INT, count_dev[i], FLOAT, &task_indx_dev[i], INT, TDD_NUM);
  	}
 	waitAll(task);
  	end_timer = my_timer();

  	printf("GPU Elapsed Time:%lf Sec.\n", end_timer - start_timer);
  
	//transfer back to host
  	for(i = 0; i < task; i++){
    		checkCudaErrors(hipMemcpyAsync(count[i], count_dev[i], n * n*sizeof(int), hipMemcpyDeviceToHost, runtime_stream));
  	}

  	checkCudaErrors(hipStreamSynchronize(runtime_stream));

  	runtime_destroy();
  	runtime_free();

	// cpu code
	start_timer = my_timer();
        for(i = 0; i < task; i++){
                get_pixelCPU(countCPU[i], task_indx[i]);
        }
        end_timer = my_timer();
        printf("CPU Elapsed Time:%lf Sec.\n", end_timer - start_timer);


  	//Determine the coloring of each pixel.
  	for(i = 0; i < task; i++)
    		det_pixel(&c_max[i], count[i]);

  	//Set the image data.
  	for(i = 0; i < task ; i++)
    		set_img(r[i], g[i], b[i], count[i], c_max[i]);

  	/*clean up*/
  	for(i = 0; i < task; i++){
    		checkCudaErrors(hipHostFree(count[i]));
    		checkCudaErrors(hipFree(count_dev[i]));
    		free(r[i]);
    		free(g[i]);
    		free(b[i]);
		free(countCPU[i]);
  	}
  	free(c_max);
  	free(r);
  	free(g);
  	free(b);
  	free(count);
	free(countCPU);
  	free(count_dev);
  	free(task_indx);
  	checkCudaErrors(hipFree(task_indx_dev));

  return 0;
}
