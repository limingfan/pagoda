#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "packet_lengths.h"
#include "headers.h"

#include "packet.h"

#define HEADER_SIZE 36
#define LEN 16384

double my_timer()
{
struct timeval time;
double _ret_val_0;
gettimeofday(( & time), 0);
_ret_val_0=(time.tv_sec+(time.tv_usec/1000000.0));
return _ret_val_0;
}

void DES_ROUND(uint32 *SK, uint32 X, uint32 Y)
{
    uint32 T;

    T = *SK ^ X;
    Y ^= SB8[ (T      ) & 0x3F ] ^
         SB6[ (T >>  8) & 0x3F ] ^
         SB4[ (T >> 16) & 0x3F ] ^
         SB2[ (T >> 24) & 0x3F ];

    T = *SK++ ^ ((X << 28) | (X >> 4));
    Y ^= SB7[ (T      ) & 0x3F ] ^
         SB5[ (T >>  8) & 0x3F ] ^
         SB3[ (T >> 16) & 0x3F ] ^
         SB1[ (T >> 24) & 0x3F ];
}
__device__ void DES_ROUND_dev(uint32 *SK, uint32 X, uint32 Y)
{
    uint32 T;

    T = *SK ^ X;
    Y ^= SB8[ (T      ) & 0x3F ] ^
         SB6[ (T >>  8) & 0x3F ] ^
         SB4[ (T >> 16) & 0x3F ] ^
         SB2[ (T >> 24) & 0x3F ];

    T = *SK++ ^ ((X << 28) | (X >> 4));
    Y ^= SB7[ (T      ) & 0x3F ] ^
         SB5[ (T >>  8) & 0x3F ] ^
         SB3[ (T >> 16) & 0x3F ] ^
         SB1[ (T >> 24) & 0x3F ];
}
/* DES key schedule */

int des_main_ks( uint32 *SK, uint8 *key )
{
    int i;
    uint32 X, Y, T;

    GET_UINT32( X, key, 0 );
    GET_UINT32( Y, key, 4 );

    /* Permuted Choice 1 */

    T =  ((Y >>  4) ^ X) & 0x0F0F0F0F;  X ^= T; Y ^= (T <<  4);
    T =  ((Y      ) ^ X) & 0x10101010;  X ^= T; Y ^= (T      );

    X =   (LHs[ (X      ) & 0xF] << 3) | (LHs[ (X >>  8) & 0xF ] << 2)
        | (LHs[ (X >> 16) & 0xF] << 1) | (LHs[ (X >> 24) & 0xF ]     )
        | (LHs[ (X >>  5) & 0xF] << 7) | (LHs[ (X >> 13) & 0xF ] << 6)
        | (LHs[ (X >> 21) & 0xF] << 5) | (LHs[ (X >> 29) & 0xF ] << 4);

    Y =   (RHs[ (Y >>  1) & 0xF] << 3) | (RHs[ (Y >>  9) & 0xF ] << 2)
        | (RHs[ (Y >> 17) & 0xF] << 1) | (RHs[ (Y >> 25) & 0xF ]     )
        | (RHs[ (Y >>  4) & 0xF] << 7) | (RHs[ (Y >> 12) & 0xF ] << 6)
        | (RHs[ (Y >> 20) & 0xF] << 5) | (RHs[ (Y >> 28) & 0xF ] << 4);

    X &= 0x0FFFFFFF;
    Y &= 0x0FFFFFFF;

    /* calculate subkeys */

    for( i = 0; i < 16; i++ )
    {
        if( i < 2 || i == 8 || i == 15 )
        {
            X = ((X <<  1) | (X >> 27)) & 0x0FFFFFFF;
            Y = ((Y <<  1) | (Y >> 27)) & 0x0FFFFFFF;
        }
        else
        {
            X = ((X <<  2) | (X >> 26)) & 0x0FFFFFFF;
            Y = ((Y <<  2) | (Y >> 26)) & 0x0FFFFFFF;
        }

        *SK++ =   ((X <<  4) & 0x24000000) | ((X << 28) & 0x10000000)
                | ((X << 14) & 0x08000000) | ((X << 18) & 0x02080000)
                | ((X <<  6) & 0x01000000) | ((X <<  9) & 0x00200000)
                | ((X >>  1) & 0x00100000) | ((X << 10) & 0x00040000)
                | ((X <<  2) & 0x00020000) | ((X >> 10) & 0x00010000)
                | ((Y >> 13) & 0x00002000) | ((Y >>  4) & 0x00001000)
                | ((Y <<  6) & 0x00000800) | ((Y >>  1) & 0x00000400)
                | ((Y >> 14) & 0x00000200) | ((Y      ) & 0x00000100)
                | ((Y >>  5) & 0x00000020) | ((Y >> 10) & 0x00000010)
                | ((Y >>  3) & 0x00000008) | ((Y >> 18) & 0x00000004)
                | ((Y >> 26) & 0x00000002) | ((Y >> 24) & 0x00000001);

        *SK++ =   ((X << 15) & 0x20000000) | ((X << 17) & 0x10000000)
                | ((X << 10) & 0x08000000) | ((X << 22) & 0x04000000)
                | ((X >>  2) & 0x02000000) | ((X <<  1) & 0x01000000)
                | ((X << 16) & 0x00200000) | ((X << 11) & 0x00100000)
                | ((X <<  3) & 0x00080000) | ((X >>  6) & 0x00040000)
                | ((X << 15) & 0x00020000) | ((X >>  4) & 0x00010000)
                | ((Y >>  2) & 0x00002000) | ((Y <<  8) & 0x00001000)
                | ((Y >> 14) & 0x00000808) | ((Y >>  9) & 0x00000400)
                | ((Y      ) & 0x00000200) | ((Y <<  7) & 0x00000100)
                | ((Y >>  7) & 0x00000020) | ((Y >>  3) & 0x00000011)
                | ((Y <<  2) & 0x00000004) | ((Y >> 21) & 0x00000002);
    }

    return( 0 );
}

int des_set_key( uint32 *esk, uint32 *dsk, uint8 key1[8],
                                uint8 key2[8], uint8 key3[8])
{
    int i;

    /* setup encryption subkeys */

    des_main_ks( esk, key1 );
    des_main_ks( dsk + 32, key2 );
    des_main_ks( esk + 64, key3 );



    /* setup decryption subkeys */

    for( i = 0; i < 32; i += 2 )
    {
        dsk[i] = esk[94 - i];
        dsk[i + 1] = esk[95 - i];
        esk[i + 32] = dsk[62 - i];
        esk[i + 33] = dsk[63 - i];
        dsk[i + 64] = esk[30 - i];
        dsk[i + 65] = esk[31 - i];

    }

    return( 0 );
}

/* DES 64-bit block encryption/decryption */

void des_crypt( uint32 *SK, uint8 *input, uint8 *output, int len)
{
    int i;
    uint32 X, Y, T;

    for(i = 0; i < len; i++){
      X = ( (uint32) *(input + i*8) << 24 )
        | ( (uint32) *(input + (i*8) + 1) << 16 )
        | ( (uint32) *(input + (i*8) + 2) <<  8 )
        | ( (uint32) *(input + (i*8) + 3)       );

      Y = ( (uint32) *(input + (i*8) + 4) << 24 )
        | ( (uint32) *(input + (i*8) + 5) << 16 )
        | ( (uint32) *(input + (i*8) + 6) <<  8 )
        | ( (uint32) *(input + (i*8) + 7)       );

      DES_IP( X, Y );

      DES_ROUND( SK, Y, X );  DES_ROUND( (SK + 2), X, Y );
      DES_ROUND( (SK + 4), Y, X );  DES_ROUND( (SK + 6), X, Y );
      DES_ROUND( (SK + 8), Y, X );  DES_ROUND( (SK + 10), X, Y );
      DES_ROUND( (SK + 12), Y, X );  DES_ROUND( (SK + 14), X, Y );
      DES_ROUND( (SK + 16), Y, X );  DES_ROUND( (SK + 18), X, Y );
      DES_ROUND( (SK + 20), Y, X );  DES_ROUND( (SK + 22), X, Y );
      DES_ROUND( (SK + 24), Y, X );  DES_ROUND( (SK + 26), X, Y );
      DES_ROUND( (SK + 28), Y, X );  DES_ROUND( (SK + 30), X, Y );

      DES_ROUND( (SK + 32), X, Y );  DES_ROUND( (SK + 34), Y, X );
      DES_ROUND( (SK + 36), X, Y );  DES_ROUND( (SK + 38), Y, X );
      DES_ROUND( (SK + 40), X, Y );  DES_ROUND( (SK + 42), Y, X );
      DES_ROUND( (SK + 44), X, Y );  DES_ROUND( (SK + 46), Y, X );
      DES_ROUND( (SK + 48), X, Y );  DES_ROUND( (SK + 50), Y, X );
      DES_ROUND( (SK + 52), X, Y );  DES_ROUND( (SK + 54), Y, X );
      DES_ROUND( (SK + 56), X, Y );  DES_ROUND( (SK + 58), Y, X );
      DES_ROUND( (SK + 60), X, Y );  DES_ROUND( (SK + 62), Y, X );

      DES_ROUND( (SK + 64), Y, X );  DES_ROUND( (SK + 66), X, Y );
      DES_ROUND( (SK + 68), Y, X );  DES_ROUND( (SK + 70), X, Y );
      DES_ROUND( (SK + 72), Y, X );  DES_ROUND( (SK + 74), X, Y );
      DES_ROUND( (SK + 76), Y, X );  DES_ROUND( (SK + 78), X, Y );
      DES_ROUND( (SK + 80), Y, X );  DES_ROUND( (SK + 82), X, Y );
      DES_ROUND( (SK + 84), Y, X );  DES_ROUND( (SK + 86), X, Y );
      DES_ROUND( (SK + 88), Y, X );  DES_ROUND( (SK + 90), X, Y );
      DES_ROUND( (SK + 92), Y, X );  DES_ROUND( (SK + 94), X, Y );

      DES_FP( Y, X );

      *(output + i*8) = (uint8) ( (Y) >> 24 );
      *(output + (i*8) + 1) = (uint8) ( (Y) >> 16 );
      *(output + (i*8) + 2) = (uint8) ( (Y) >>  8 );
      *(output + (i*8) + 3) = (uint8) ( (Y)       );
      *(output + (i*8) + 4) = (uint8) ( (X) >> 24 );
      *(output + (i*8) + 5) = (uint8) ( (X) >> 16 );
      *(output + (i*8) + 6) = (uint8) ( (X) >>  8 );
      *(output + (i*8) + 7) = (uint8) ( (X)       );
	
    }
}

__device__ void des_crypt_dev( uint32 *SK, uint8 *input, uint8 *output, int len, int TD_NUM)
{
    int i;
    uint32 X, Y, T;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < TD_NUM){
      for(i = 0; i < len/TD_NUM; i++){
      
	X = ( (uint32) *(input + (i*TD_NUM+tid)*8) << 24 )
        | ( (uint32) *(input + ((i*TD_NUM+tid)*8) + 1) << 16 )
        | ( (uint32) *(input + ((i*TD_NUM+tid)*8) + 2) <<  8 )
        | ( (uint32) *(input + ((i*TD_NUM+tid)*8) + 3)       );

        Y = ( (uint32) *(input + ((i*TD_NUM+tid)*8) + 4) << 24 )
        | ( (uint32) *(input + ((i*TD_NUM+tid)*8) + 5) << 16 )
        | ( (uint32) *(input + ((i*TD_NUM+tid)*8) + 6) <<  8 )
        | ( (uint32) *(input + ((i*TD_NUM+tid)*8) + 7)       );


        DES_IP( X, Y );

        DES_ROUND_dev( SK, Y, X );  DES_ROUND_dev( (SK + 2), X, Y );
        DES_ROUND_dev( (SK + 4), Y, X );  DES_ROUND_dev( (SK + 6), X, Y );
        DES_ROUND_dev( (SK + 8), Y, X );  DES_ROUND_dev( (SK + 10), X, Y );
        DES_ROUND_dev( (SK + 12), Y, X );  DES_ROUND_dev( (SK + 14), X, Y );
        DES_ROUND_dev( (SK + 16), Y, X );  DES_ROUND_dev( (SK + 18), X, Y );
        DES_ROUND_dev( (SK + 20), Y, X );  DES_ROUND_dev( (SK + 22), X, Y );
        DES_ROUND_dev( (SK + 24), Y, X );  DES_ROUND_dev( (SK + 26), X, Y );
        DES_ROUND_dev( (SK + 28), Y, X );  DES_ROUND_dev( (SK + 30), X, Y );

        DES_ROUND_dev( (SK + 32), X, Y );  DES_ROUND_dev( (SK + 34), Y, X );
        DES_ROUND_dev( (SK + 36), X, Y );  DES_ROUND_dev( (SK + 38), Y, X );
        DES_ROUND_dev( (SK + 40), X, Y );  DES_ROUND_dev( (SK + 42), Y, X );
        DES_ROUND_dev( (SK + 44), X, Y );  DES_ROUND_dev( (SK + 46), Y, X );
        DES_ROUND_dev( (SK + 48), X, Y );  DES_ROUND_dev( (SK + 50), Y, X );
        DES_ROUND_dev( (SK + 52), X, Y );  DES_ROUND_dev( (SK + 54), Y, X );
        DES_ROUND_dev( (SK + 56), X, Y );  DES_ROUND_dev( (SK + 58), Y, X );
        DES_ROUND_dev( (SK + 60), X, Y );  DES_ROUND_dev( (SK + 62), Y, X );

        DES_ROUND_dev( (SK + 64), Y, X );  DES_ROUND_dev( (SK + 66), X, Y );
        DES_ROUND_dev( (SK + 68), Y, X );  DES_ROUND_dev( (SK + 70), X, Y );
        DES_ROUND_dev( (SK + 72), Y, X );  DES_ROUND_dev( (SK + 74), X, Y );
        DES_ROUND_dev( (SK + 76), Y, X );  DES_ROUND_dev( (SK + 78), X, Y );
        DES_ROUND_dev( (SK + 80), Y, X );  DES_ROUND_dev( (SK + 82), X, Y );
        DES_ROUND_dev( (SK + 84), Y, X );  DES_ROUND_dev( (SK + 86), X, Y );
        DES_ROUND_dev( (SK + 88), Y, X );  DES_ROUND_dev( (SK + 90), X, Y );
        DES_ROUND_dev( (SK + 92), Y, X );  DES_ROUND_dev( (SK + 94), X, Y );

        DES_FP( Y, X );

      *(output + (i*TD_NUM+tid)*8) = (uint8) ( (Y) >> 24 );
      *(output + ((i*TD_NUM+tid)*8) + 1) = (uint8) ( (Y) >> 16 );
      *(output + ((i*TD_NUM+tid)*8) + 2) = (uint8) ( (Y) >>  8 );
      *(output + ((i*TD_NUM+tid)*8) + 3) = (uint8) ( (Y)       );
      *(output + ((i*TD_NUM+tid)*8) + 4) = (uint8) ( (X) >> 24 );
      *(output + ((i*TD_NUM+tid)*8) + 5) = (uint8) ( (X) >> 16 );
      *(output + ((i*TD_NUM+tid)*8) + 6) = (uint8) ( (X) >>  8 );
      *(output + ((i*TD_NUM+tid)*8) + 7) = (uint8) ( (X)       );

      }
    }
}
__global__ void des_encrypt_dev( uint32 *esk, uint32 *dsk, uint8 *input, uint8 *output, int len, int TD_NUM)
{
    des_crypt_dev( esk, input, input, len, TD_NUM);
    des_crypt_dev( dsk, input, output, len, TD_NUM);
}

void des_encrypt( uint32 *esk, uint32 *dsk, uint8 *input, uint8 *output, int len)
{
    des_crypt( esk, input, input, len);
    des_crypt( dsk, input, output, len);
}


/* For debugging purposes, to get the size of the packet_numberth packet */
unsigned int
packet_size (unsigned int packet_number)
{
  packet_number = packet_number % MAX_INDEX;
  return (packet_lengths[packet_number]);
}


int main (int argc, char **argv)
{
  	int i, j;
  	unsigned char **packet_in, **packet_in_dev, 
		**packet_out, **packet_out_dev, **packet_open;
  	int *packet_length;

  	uint32 *des_esk;
  	uint32 *des_dsk;

  	uint32 *des_esk_dev;
  	uint32 *des_dsk_dev;

	if(argc < 3){
                printf("Error input option:./des #thread #task\n");
                exit(1);
        }

	int TD_NUM = atoi(argv[1]);
	int numpackets = atoi(argv[2]);

	printf("CUDA baseline DES:#thread:%d, #task:%d\n", TD_NUM, numpackets);

  	hipStream_t des_stream[numpackets];

  	setenv("CUDA_DEVICE_MAX_CONNECTIONS", "32", 1);

  	for(i = 0; i < numpackets; i++){
    		checkCudaErrors(hipStreamCreate(&des_stream[i]));
  	}


  	double start_timer, end_timer;
 
  	packet_in = (unsigned char**)malloc(numpackets*sizeof(unsigned char*)); 
  	packet_in_dev = (unsigned char**)malloc(numpackets*sizeof(unsigned char*));
  	packet_out = (unsigned char**)malloc(numpackets*sizeof(unsigned char*)); 
  	packet_out_dev = (unsigned char**)malloc(numpackets*sizeof(unsigned char*));
  	packet_open = (unsigned char**)malloc(numpackets*sizeof(unsigned char*));
  	packet_length = (int*)malloc(numpackets*sizeof(int));

  	checkCudaErrors(hipHostAlloc(&des_esk, 96*sizeof(uint32), hipHostMallocDefault));
 	checkCudaErrors(hipMalloc(&des_esk_dev, 96*sizeof(uint32)));
  	checkCudaErrors(hipHostAlloc(&des_dsk, 96*sizeof(uint32), hipHostMallocDefault));
  	checkCudaErrors(hipMalloc(&des_dsk_dev, 96*sizeof(uint32)));

  	/*Generate encryption key*/
  	des_set_key(des_esk, des_dsk, DES3_keys[0], DES3_keys[1], DES3_keys[2]);
  	//memory allocation for packet
  	for(i = 0; i < numpackets; i++){
      		packet_length[i] = LEN;
      		//packet_length[i] = packet_size(i);
      		checkCudaErrors(hipHostAlloc(&packet_in[i], packet_length[i]*sizeof(unsigned char), hipHostMallocDefault));
      		checkCudaErrors(hipMalloc(&packet_in_dev[i], packet_length[i]*sizeof(unsigned char)));
      		checkCudaErrors(hipHostAlloc(&packet_out[i], packet_length[i]*sizeof(unsigned char), hipHostMallocDefault));
      		checkCudaErrors(hipMalloc(&packet_out_dev[i], packet_length[i]*sizeof(unsigned char)));
      		packet_open[i] =  (unsigned char *) malloc (packet_length[i]);
  	}
  
  	//generate packet
  	for(i = 0; i < numpackets; i++){
      		for(j = 0; j < packet_length[i]; j++){
          		if(j < HEADER_SIZE ){
              			packet_in[i][j] = headers[i % MAX_PACKETS][j];
          		}else{
              			packet_in[i][j] = DES3_init[j%8];
          		}
      		}
  	}

  	// copy data to GPU
  	for(i = 0; i < numpackets; i++){
     		checkCudaErrors(hipMemcpyAsync(packet_in_dev[i], packet_in[i], packet_length[i]*sizeof(unsigned char), hipMemcpyHostToDevice, des_stream[i%32]));
  	}
  	checkCudaErrors(hipMemcpyAsync(des_esk_dev, des_esk, 96*sizeof(uint32), hipMemcpyHostToDevice, des_stream[0]));
  	checkCudaErrors(hipMemcpyAsync(des_dsk_dev, des_dsk, 96*sizeof(uint32), hipMemcpyHostToDevice, des_stream[0]));
  	checkCudaErrors(hipDeviceSynchronize());

  	start_timer = my_timer();
  	// run des
  	for(i = 0; i < numpackets; i++){
		des_encrypt_dev<<<1, TD_NUM, 0, des_stream[i%32]>>>( des_esk_dev, des_esk_dev, packet_in_dev[i], 
					packet_out_dev[i], packet_length[i]/8, TD_NUM);

  	}
  	checkCudaErrors(hipDeviceSynchronize());
  	end_timer = my_timer();
  	printf("GPU time:%lf Sec.\n", end_timer - start_timer);

  	for(i = 0; i < numpackets; i++){
     		checkCudaErrors(hipMemcpyAsync(packet_out[i], packet_out_dev[i], packet_length[i]*sizeof(unsigned char), hipMemcpyDeviceToHost, des_stream[i%32]));
  	}
  	checkCudaErrors(hipDeviceSynchronize());
  	start_timer = my_timer();
  	// run des
  	for(i = 0; i < numpackets; i++){
        	des_encrypt(des_esk, des_dsk, packet_in[i], packet_open[i], packet_length[i]/8);
		//des_crypt( des_esk, packet_in[i], packet_open[i], packet_length[i]/8);

  	}
  	end_timer = my_timer();
  	printf("CPU time:%lf Sec.\n", end_timer - start_timer);

	/*Verification*/
	long long flag = 0;
        long long lenPacket = 0;
        printf("Verifying\n");
  	for(i = 0; i < numpackets; i++){
    		for(j = 0; j < packet_length[i]; j++){
      			if(packet_out[i][j] != packet_open[i][j]){
        			printf("Error:%u, %u, %d, %d\n", packet_out[i][j], packet_open[i][j], i, j);
        			break;
      			}
			flag ++;
    		}
		lenPacket += packet_length[i];
  	}
	if(flag == lenPacket) printf("Verify Successfully\n");
 
  	for(i = 0; i < numpackets; i++){
    		checkCudaErrors(hipStreamDestroy(des_stream[i]));
    		checkCudaErrors(hipHostFree(packet_in[i]));
    		checkCudaErrors(hipFree(packet_in_dev[i]));
    		checkCudaErrors(hipHostFree(packet_out[i]));
    		checkCudaErrors(hipFree(packet_out_dev[i]));
    		free(packet_open[i]);
  	}

  	checkCudaErrors(hipHostFree(des_esk));
  	checkCudaErrors(hipFree(des_esk_dev));
  	checkCudaErrors(hipHostFree(des_dsk));
  	checkCudaErrors(hipFree(des_dsk_dev));


 	free(packet_in);
  	free(packet_in_dev);
  	free(packet_out);
  	free(packet_out_dev);
  	free(packet_open);
  	free(packet_length);

  	if(hipDeviceReset()== hipSuccess) printf("Reset successful\n");

  
  return 0;
}
