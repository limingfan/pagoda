#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "kernel.h"
#include "headers.h"
#include "runtime.cuh"

#define TK_NUM 2048 //num. of task in each category
#define task (TK_NUM*4)

#define MAX_PACKETS     100
#define MAX_INDEX       32768  

double my_timer()
{
struct timeval time;
double _ret_val_0;
gettimeofday(( & time), 0);
_ret_val_0=(time.tv_sec+(time.tv_usec/1000000.0));
return _ret_val_0;
}

void init_matrix(int **A, int **B, int **C, int **D);
void init_filter(float **r, float **Vect_Up, float **Vect_F, 
                float **Vect_H, float **H, float **F, float *y, float **Vect_F_host);
void init_des(unsigned char **packet_in);

int des_main_ks( uint32 *SK, uint8 *key );
int des_set_key( uint32 *esk, uint32 *dsk, uint8 key1[8],
                                uint8 key2[8], uint8 key3[8]);


int main(){

  	int i, j;
  	int *h_A[TK_NUM], *h_B[TK_NUM], *h_C[TK_NUM], *h_D[TK_NUM];
  	int *d_A[TK_NUM], *d_B[TK_NUM], *d_C[TK_NUM];

  	int *h_count[TK_NUM];
  	int *d_count[TK_NUM];
  	int *h_count_host[TK_NUM];
  	float *h_task_indx;
  	float *d_task_indx;

  	float *h_r[TK_NUM],*d_r[TK_NUM];
  	float *y, *h_H[TK_NUM], *d_H[TK_NUM];
  	float *h_F[TK_NUM], *d_F[TK_NUM];

  	float *h_Vect_H[TK_NUM], *d_Vect_H[TK_NUM]; // output of the F
  	float *h_Vect_Dn[TK_NUM], *d_Vect_Dn[TK_NUM]; // output of the down sampler
  	float *h_Vect_Up[TK_NUM], *d_Vect_Up[TK_NUM]; // output of the up sampler
  	float *h_Vect_F[TK_NUM], *d_Vect_F[TK_NUM], *h_Vect_F_host[TK_NUM]; // this is the output of the

  	unsigned char *h_packet_in[TK_NUM], *d_packet_in[TK_NUM];
  	unsigned char *h_packet_out[TK_NUM], *d_packet_out[TK_NUM];
  	unsigned char *h_packet_host[TK_NUM];

  	uint32 *h_des_esk;
  	uint32 *h_des_dsk;

  	uint32 *d_des_esk;
  	uint32 *d_des_dsk;

  	double start_timer, end_timer;

  	//matrix mult.
  	for(i = 0; i < TK_NUM; i++){
    		checkCudaErrors(hipHostAlloc(&h_A[i], MSIZE*sizeof(int), hipHostMallocDefault));
    		checkCudaErrors(hipHostAlloc(&h_B[i], MSIZE*sizeof(int), hipHostMallocDefault));
    		checkCudaErrors(hipHostAlloc(&h_C[i], MSIZE*sizeof(int), hipHostMallocDefault));
  	}

  	for(i = 0; i < TK_NUM; i++){
    		checkCudaErrors(hipMalloc(&d_A[i], MSIZE*sizeof(int)));
    		checkCudaErrors(hipMalloc(&d_B[i], MSIZE*sizeof(int)));
    		checkCudaErrors(hipMalloc(&d_C[i], MSIZE*sizeof(int)));
    		h_D[i] = (int*)malloc(sizeof(int)*MSIZE);
  	}

  	// mandelbrot
  	h_task_indx = (float*)malloc(TK_NUM * sizeof(float));
  	checkCudaErrors(hipMalloc(&d_task_indx, TK_NUM *sizeof(float)));

  	for(i = 0; i < TK_NUM; i++) {
    		h_task_indx[i] = (float)(i/(TK_NUM/2.0));
    		checkCudaErrors(hipHostAlloc(&h_count[i], n * n *sizeof(int), NULL));
    		checkCudaErrors(hipMalloc(&d_count[i], n * n *sizeof(int)));
    		h_count_host[i] = (int*)malloc(n * n * sizeof(int));

  	}

  	//filter bank
  	for(i = 0; i < TK_NUM; i++){
    		checkCudaErrors(hipHostAlloc(&h_r[i], N_sim*sizeof(float), hipHostMallocDefault));
    		checkCudaErrors(hipMalloc(&d_r[i], N_sim*sizeof(float)));
    		checkCudaErrors(hipHostAlloc(&h_H[i], N_col*sizeof(float), hipHostMallocDefault));
    		checkCudaErrors(hipMalloc(&d_H[i], N_col*sizeof(float)));
    		checkCudaErrors(hipHostAlloc(&h_F[i], N_col*sizeof(float), hipHostMallocDefault));
    		checkCudaErrors(hipMalloc(&d_F[i], N_col*sizeof(float)));

    		checkCudaErrors(hipHostAlloc(&h_Vect_H[i], N_sim*sizeof(float), hipHostMallocDefault));
    		checkCudaErrors(hipMalloc(&d_Vect_H[i], N_sim*sizeof(float)));
    		checkCudaErrors(hipHostAlloc(&h_Vect_Dn[i], (N_sim/N_samp)*sizeof(float), hipHostMallocDefault));
    		checkCudaErrors(hipMalloc(&d_Vect_Dn[i], (N_sim/N_samp)*sizeof(float)));
    		checkCudaErrors(hipHostAlloc(&h_Vect_Up[i], N_sim*sizeof(float), hipHostMallocDefault));
    		checkCudaErrors(hipMalloc(&d_Vect_Up[i], N_sim*sizeof(float)));
    		checkCudaErrors(hipHostAlloc(&h_Vect_F[i], N_sim*sizeof(float), hipHostMallocDefault));
    		checkCudaErrors(hipMalloc(&d_Vect_F[i], N_sim*sizeof(float)));
    		h_Vect_F_host[i] = (float*)malloc(N_sim*sizeof(float));
  	}
  	y = (float*)malloc(N_sim*sizeof(float));

  	//DES
  	for(i = 0; i < TK_NUM; i++){
      		checkCudaErrors(hipHostAlloc(&h_packet_in[i], LEN*sizeof(unsigned char), hipHostMallocDefault));
      		checkCudaErrors(hipMalloc(&d_packet_in[i], LEN*sizeof(unsigned char)));
      		checkCudaErrors(hipHostAlloc(&h_packet_out[i], LEN*sizeof(unsigned char), hipHostMallocDefault));
      		checkCudaErrors(hipMalloc(&d_packet_out[i], LEN*sizeof(unsigned char)));
      		h_packet_host[i] =  (unsigned char *) malloc (LEN*sizeof(unsigned char));
  	}

  	checkCudaErrors(hipHostAlloc(&h_des_esk, 96*sizeof(uint32), hipHostMallocDefault));
  	checkCudaErrors(hipMalloc(&d_des_esk, 96*sizeof(uint32)));
  	checkCudaErrors(hipHostAlloc(&h_des_dsk, 96*sizeof(uint32), hipHostMallocDefault));
  	checkCudaErrors(hipMalloc(&d_des_dsk, 96*sizeof(uint32)));


   	/*Generate encryption key*/
  	des_set_key(h_des_esk, h_des_dsk, DES3_keys[0], DES3_keys[1], DES3_keys[2]);
  

  	//Init.matrix
  	init_matrix(h_A, h_B, h_C, h_D);
  	//Init filter
  	init_filter(h_r, h_Vect_Up, h_Vect_F, 
                h_Vect_H, h_H, h_F, y, h_Vect_F_host);
  	//Init DES
  	init_des(h_packet_in);

  	//Init runtime
  	runtime_init();

  	double timer = 0.0;

  	start_timer = my_timer();

  	//mem copy
  	for(i = 0; i < TK_NUM; i++){
    		checkCudaErrors(hipMemcpyAsync(d_A[i], h_A[i], MSIZE*sizeof(int), hipMemcpyHostToDevice, runtime_stream));
    		checkCudaErrors(hipMemcpyAsync(d_B[i], h_B[i], MSIZE*sizeof(int), hipMemcpyHostToDevice, runtime_stream));

    		checkCudaErrors(hipMemcpyAsync(d_r[i], h_r[i], N_sim*sizeof(float), hipMemcpyHostToDevice, runtime_stream));
    		checkCudaErrors(hipMemcpyAsync(d_Vect_Up[i], h_Vect_Up[i], N_sim*sizeof(float), hipMemcpyHostToDevice, runtime_stream));
    		checkCudaErrors(hipMemcpyAsync(d_Vect_F[i], h_Vect_F[i], N_sim*sizeof(float), hipMemcpyHostToDevice, runtime_stream));
    		checkCudaErrors(hipMemcpyAsync(d_Vect_H[i], h_Vect_H[i], N_sim*sizeof(float), hipMemcpyHostToDevice, runtime_stream));
    		checkCudaErrors(hipMemcpyAsync(d_H[i], h_H[i], N_col*sizeof(float), hipMemcpyHostToDevice, runtime_stream));
    		checkCudaErrors(hipMemcpyAsync(d_F[i], h_F[i], N_col*sizeof(float), hipMemcpyHostToDevice, runtime_stream));
    		checkCudaErrors(hipMemcpyAsync(d_packet_in[i], h_packet_in[i], LEN*sizeof(unsigned char), hipMemcpyHostToDevice, runtime_stream));

  	}

  	checkCudaErrors(hipMemcpyAsync(d_task_indx, h_task_indx, TK_NUM*sizeof(float), hipMemcpyHostToDevice, runtime_stream));
  	checkCudaErrors(hipMemcpyAsync(d_des_esk, h_des_esk, 96*sizeof(uint32), hipMemcpyHostToDevice, runtime_stream));
  	checkCudaErrors(hipMemcpyAsync(d_des_dsk, h_des_dsk, 96*sizeof(uint32), hipMemcpyHostToDevice, runtime_stream));
  	checkCudaErrors(hipStreamSynchronize(runtime_stream));
  	end_timer = my_timer();
  	timer = end_timer - start_timer;
  
  	printf("Pagoda MultiWrok: #task:%d, #thread:%d\n", task, TDD_NUM);
  	// compute
  	int mult_c, mand_c, filter_c, des_c;
  	mult_c = 0, mand_c = 0, filter_c = 0, des_c = 0;
  	start_timer = my_timer();

  	for(i = 0; i < task; i++){
    		switch(i%4){
      			case 0:
        			taskLaunch(9, INT, TDD_NUM, INT, 1, INT, 0, INT, 0, INT, 0, INT, d_A[mult_c], INT, d_B[mult_c], INT, d_C[mult_c], INT, MROW);
        			mult_c ++;
        			break;
      			case 1:
        			taskLaunch(7, INT, TDD_NUM, INT, 1, INT, 0, INT, 0, INT, 1, INT, d_count[mand_c], INT, &d_task_indx[mand_c]);
        			mand_c ++;
        			break;
      			case 2:
        			taskLaunch(12, INT, TDD_NUM, INT, 1, INT, 0, INT, 1, INT, 2, FLOAT, d_r[filter_c],
                			FLOAT, d_H[filter_c], FLOAT, d_Vect_H[filter_c], FLOAT, d_Vect_Dn[filter_c], FLOAT, d_Vect_Up[filter_c],
                			FLOAT, d_Vect_F[filter_c], FLOAT, d_F[filter_c]);
        				filter_c ++;
        			break;
      			case 3:
        			taskLaunch(10, INT, TDD_NUM, INT, 1, INT, 0, INT, 0, INT, 3, INT32, d_des_esk, INT32, d_des_esk,
                               		CHAR, d_packet_in[des_c], CHAR, d_packet_out[des_c], INT, LEN/8);

        			des_c ++;
        			break;
    
    		}
  	}
  	waitAll(task);

  	end_timer = my_timer();
  	printf("GPU elapsed time:%lf Sec.\n", end_timer - start_timer);
 
  	start_timer = my_timer(); 

  	// memory copy back
  	for(i = 0; i < TK_NUM; i++){
    		checkCudaErrors(hipMemcpyAsync(h_C[i],d_C[i], MSIZE*sizeof(int), hipMemcpyDeviceToHost, runtime_stream));
    		checkCudaErrors(hipMemcpyAsync(h_count[i], d_count[i], n * n*sizeof(int), hipMemcpyDeviceToHost, runtime_stream));
    		checkCudaErrors(hipMemcpyAsync(h_Vect_F[i], d_Vect_F[i], N_sim*sizeof(float), hipMemcpyDeviceToHost, runtime_stream));
    		checkCudaErrors(hipMemcpyAsync(h_packet_out[i], d_packet_out[i], LEN*sizeof(unsigned char), hipMemcpyDeviceToHost, runtime_stream));
  	}
  	checkCudaErrors(hipStreamSynchronize(runtime_stream));
  	end_timer = my_timer();
  	timer += (end_timer - start_timer);
  	printf("Mem. copy time:%lf Sec.\n", timer);

  	runtime_destroy();
  	runtime_free();

  	mult_c = 0, mand_c = 0, filter_c = 0, des_c = 0;

  	start_timer = my_timer();

  	// cpu compute
  	for(i = 0; i < task; i++){
    		switch(i%4){
      			case 0:
        			mult(h_A[mult_c], h_B[mult_c], h_D[mult_c], MROW);
        			mult_c ++;
        			break;
      			case 1:
        			h_get_pixel(h_count_host[mand_c], h_task_indx[mand_c]);
        			mand_c++;
        			break;

      			case 2:
        			h_FBCore(h_r[filter_c], h_H[filter_c], h_Vect_H[filter_c], 
				h_Vect_Dn[filter_c], h_Vect_Up[filter_c], h_Vect_F_host[filter_c], h_F[filter_c]);
        			filter_c ++;
        			break;

      			case 3:
        			des_encrypt(h_des_esk, h_des_dsk, h_packet_in[des_c], h_packet_host[des_c], LEN/8);
        			des_c ++; 
        			break; 
    		}
  	}
  	end_timer = my_timer();
  	printf("CPU elapsed time:%lf Sec.\n", end_timer - start_timer);


  	//verificiation
  	printf("Verifying\n");
	long long flag = 0;

  	for(i = 0; i < TK_NUM; i++){
    		for(j = 0; j < MSIZE; j++){
      			if(h_C[i][j] != h_D[i][j]){
        			printf("Mult, Error:%d, %d\n", h_C[i][j], h_D[i][j]);
       				 break;
      			}
			flag ++;
    		}

    		for(j = 0; j < N_sim; j++){
      			if(abs(h_Vect_F[i][j]- h_Vect_F_host[i][j]) > 0.1){
        			printf("Filter Error:%f, %f\n", h_Vect_F[i][j], h_Vect_F_host[i][j], i, j);
        			break;
      			}
			flag ++;
    		}

    		for(j = 0; j < LEN; j++){
        		if(h_packet_out[i][j] != h_packet_host[i][j]){
          			printf("DES Error:%u, %u, %d, %d\n", h_packet_out[i][j], h_packet_host[i][j], i, j);
          			break;
      			}
			flag ++;
    		}

  	}
	if(flag == (TK_NUM * MSIZE + TK_NUM * N_sim + TK_NUM * LEN)) printf("Verifying Successfully\n");

  	//free mem.
  	for(i = 0; i < TK_NUM; i++){

    		checkCudaErrors(hipHostFree(h_A[i]));
    		checkCudaErrors(hipFree(d_A[i]));
    		checkCudaErrors(hipHostFree(h_B[i]));
    		checkCudaErrors(hipFree(d_B[i]));
   	 	checkCudaErrors(hipHostFree(h_C[i]));
    		checkCudaErrors(hipFree(d_C[i]));

    		checkCudaErrors(hipHostFree(h_count[i]));
    		checkCudaErrors(hipFree(d_count[i]));

    		checkCudaErrors(hipHostFree(h_r[i]));
    		checkCudaErrors(hipFree(d_r[i]));
    		checkCudaErrors(hipHostFree(h_H[i]));
    		checkCudaErrors(hipFree(d_H[i]));
    		checkCudaErrors(hipHostFree(h_F[i]));
    		checkCudaErrors(hipFree(d_F[i]));

    		checkCudaErrors(hipHostFree(h_Vect_H[i]));
    		checkCudaErrors(hipFree(d_Vect_H[i]));
    		checkCudaErrors(hipHostFree(h_Vect_Dn[i]));
    		checkCudaErrors(hipFree(d_Vect_Dn[i]));
    		checkCudaErrors(hipHostFree(h_Vect_Up[i]));
    		checkCudaErrors(hipFree(d_Vect_Up[i]));
    		checkCudaErrors(hipHostFree(h_Vect_F[i]));
    		checkCudaErrors(hipFree(d_Vect_F[i]));

    		checkCudaErrors(hipHostFree(h_packet_in[i]));
    		checkCudaErrors(hipFree(d_packet_in[i]));
    		checkCudaErrors(hipHostFree(h_packet_out[i]));
    		checkCudaErrors(hipFree(d_packet_out[i]));

    		free(h_packet_host[i]);
    		free(h_count_host[i]);
    		free(h_Vect_F_host[i]);
  	}

  	checkCudaErrors(hipFree(d_task_indx));

  	checkCudaErrors(hipHostFree(h_des_esk));
  	checkCudaErrors(hipFree(d_des_esk));
  	checkCudaErrors(hipHostFree(h_des_dsk));
  	checkCudaErrors(hipFree(d_des_dsk));

  	free(h_task_indx);
  	free(y);

  	if(hipDeviceReset()== hipSuccess) printf("Reset successful\n");

return 0;
}

void init_matrix(int **A, int **B, int **C, int **D){
  int i, j;

  for(i = 0; i < TK_NUM; i++){
    for(j = 0; j < MSIZE; j++){
      A[i][j] = (i%MROW)+1;
      B[i][j] = (i%MCOL)+1;
      C[i][j] = 0;
      D[i][j] = 0;
    }
  }
}

void init_filter(float **r, float **Vect_Up, float **Vect_F, 
		float **Vect_H, float **H, float **F, float *y, float **Vect_F_host){
  int i, j;

  for(i = 0; i < TK_NUM; i++)
    for(j = 0; j < N_sim; j++){
      r[i][j] = j + 0.0001;
      y[j] = 0;
      Vect_Up[i][j] = 0;
      Vect_F[i][j] = 0;
      Vect_H[i][j]=0;
      Vect_F_host[i][j] = 0;
    }

  for(i = 0; i < TK_NUM; i++)
    for(j = 0; j < N_col; j++){
      H[i][j] = 0.0001;
      F[i][j] = 0.0001;
    }

}

void init_des(unsigned char **packet_in){
  int i, j;
  for(i = 0; i < TK_NUM; i++){
      for(j = 0; j < LEN; j++){
          if(j < HEADER_SIZE ){
              packet_in[i][j] = headers[i % MAX_PACKETS][j];
          }else{
              packet_in[i][j] = DES3_init[j%8];
          }
      }
  }
}

