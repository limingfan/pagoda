#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "../../common/para.h"
#define BLOCK_SIZE 8
#define task (TK_NUM * BT_NUM)

#define C_norm  (0.3535533905932737) // 1 / (8^0.5)
#define C_a  	(1.387039845322148) //!< a = (2^0.5) * cos(    pi / 16);  Used in forward and inverse DCT.
#define C_b 	(1.306562964876377) //!< b = (2^0.5) * cos(    pi /  8);  Used in forward and inverse DCT.
#define C_c  	(1.175875602419359) //!< c = (2^0.5) * cos(3 * pi / 16);  Used in forward and inverse DCT.
#define C_d     (0.785694958387102) //!< d = (2^0.5) * cos(5 * pi / 16);  Used in forward and inverse DCT.
#define C_e     (0.541196100146197) //!< e = (2^0.5) * cos(3 * pi /  8);  Used in forward and inverse DCT.
#define C_f 	(0.275899379282943) //!< f = (2^0.5) * cos(7 * pi / 16);  Used in forward and inverse DCT.


double my_timer()
{
struct timeval time;
double _ret_val_0;
gettimeofday(( & time), 0);
_ret_val_0=(time.tv_sec+(time.tv_usec/1000000.0));
return _ret_val_0;
}

void DCT(float *fSrc, float *fDst, int *Stride, int *size, int *thread, int index);
__global__ void d_DCT(float *fSrc, float *fDst, int *Stride, int *size, int *thread, int index);
void SubroutineDCTvector(float *FirstIn, int StepIn, float *FirstOut, int StepOut);
void SubroutineIDCTvector(float *FirstIn, int StepIn, float *FirstOut, int StepOut);
void computeIDCT(float *fSrc, float *fDst, int *Stride, int *size, int *thread, int index);
void computeDCT(float *fSrc, float *fDst, int *Stride, int *size, int *thread, int index);
__device__ void d_SubroutineDCTvector(float *FirstIn, int StepIn, float *FirstOut, int StepOut);
__device__ void d_SubroutineIDCTvector(float *FirstIn, int StepIn, float *FirstOut, int StepOut);
__device__ void d_computeDCT(float *fSrc, float *fDst, int *Stride, int *size, int *thread, int index);
__device__ void d_computeIDCT(float *fSrc, float *fDst, int *Stride, int *size, int *thread, int index);

int main(){
  	int i, j;
  	float *A[BT_NUM], *C[BT_NUM], *D[BT_NUM];
  	float *A_dev[BT_NUM], *C_dev[BT_NUM];
  	double start_timer, end_timer;
  	int num_thread[task], *num_thread_dev;
  	int num_size[BT_NUM];
  	int pos_task[BT_NUM][TK_NUM];
  	int *pos_task_dev[BT_NUM];
  	int Stride[BT_NUM][TK_NUM], *d_Stride[BT_NUM];
	hipSetDevice(0);
  	FILE *fp;

  	fp = fopen("rand.txt", "r");
  	for(i = 0; i < task; i++)
    		fscanf(fp, "%1d", &num_thread[i]);

  	fclose(fp);

  	for(i = 0; i < task; i++){
    		if(num_thread[i] == 1){
			num_thread[i] = 64;
    		}else{
    			num_thread[i] *= 32;
    		}
  	}

  	for(i = 0; i < BT_NUM; i++){
    		num_size[i] = 0;
  	}

  	for(i = 0; i < BT_NUM; i++){
    		for(j = 0; j < TK_NUM; j++){
        		num_size[i] += (num_thread[i*TK_NUM+j])*
                        	(num_thread[i*TK_NUM+j]);
        		Stride[i][j] = ((int)ceil((num_thread[i*TK_NUM+j]*sizeof(float))/16.0f))*16 / sizeof(float);
        		pos_task[i][j] = 0;
        		if(j > 0) pos_task[i][j] += pos_task[i][j-1] + (num_thread[i*TK_NUM+j-1])*
                        	(num_thread[i*TK_NUM+j-1]);

    		}
  	}

  	for(i = 0; i < BT_NUM; i++){
    		checkCudaErrors(hipHostAlloc(&A[i], num_size[i]*sizeof(float), hipHostMallocDefault));
    		checkCudaErrors(hipMalloc(&A_dev[i], num_size[i]*sizeof(float)));
    		checkCudaErrors(hipHostAlloc(&C[i], num_size[i]*sizeof(float), hipHostMallocDefault));
    		checkCudaErrors(hipMalloc(&C_dev[i], num_size[i]*sizeof(float)));
    		D[i] = (float*)malloc(sizeof(float)*num_size[i]);
    		checkCudaErrors(hipMalloc(&pos_task_dev[i], TK_NUM*sizeof(int)));
    		checkCudaErrors(hipMalloc(&d_Stride[i], TK_NUM*sizeof(int)));
  	}

  	checkCudaErrors(hipMalloc(&num_thread_dev, task*sizeof(int)));

	printf("DCT inputs are generating\n");
  	// Init matrix
  	for(i = 0; i < BT_NUM; i++){
    		for(j = 0; j < num_size[i]; j++){
      			A[i][j] = (i%num_size[i])+1;
      			C[i][j] = 0;
      			D[i][j] = 0;
    		}
  	}

  	//transfer data to device
  	for(i = 0; i < BT_NUM; i++){
    		checkCudaErrors(hipMemcpy(A_dev[i], A[i], num_size[i]*sizeof(float), hipMemcpyHostToDevice));
    		checkCudaErrors(hipMemcpy(pos_task_dev[i], pos_task[i], TK_NUM*sizeof(int), hipMemcpyHostToDevice));
    		checkCudaErrors(hipMemcpy(d_Stride[i], Stride[i], TK_NUM*sizeof(int), hipMemcpyHostToDevice));

  	}
  	checkCudaErrors(hipMemcpy(num_thread_dev, num_thread, task*sizeof(int), hipMemcpyHostToDevice));
  	checkCudaErrors(hipDeviceSynchronize());
	printf("DCT CUDA static fusion is running\n");
  	start_timer = my_timer();
  	for(i = 0; i < BT_NUM; i++){
    		d_DCT<<<TK_NUM, TDK_NUM>>>(A_dev[i], C_dev[i], d_Stride[i], pos_task_dev[i], num_thread_dev, i);
  	}
  	checkCudaErrors(hipDeviceSynchronize());
  	end_timer = my_timer();
  	printf("DCT CUDA static fusion Elapsed Time: %lf Sec.\n", end_timer - start_timer);

  	//transfer data back to host
  	for(i = 0; i < BT_NUM; i++)
    		checkCudaErrors(hipMemcpy(C[i], C_dev[i], num_size[i]*sizeof(float), hipMemcpyDeviceToHost));
  	checkCudaErrors(hipDeviceSynchronize());

	printf("CPU program running\n");
  	start_timer = my_timer();
  	for(i = 0; i < BT_NUM; i++){
    		DCT(A[i], D[i], Stride[i], pos_task[i], num_thread, i);
  	}
  	end_timer = my_timer();
  	//printf("The CPU Elapsed Time:%lf Sec.\n", end_timer - start_timer);


  	//Verification
	printf("verifying\n");
	int flag = 0;
  	for(i = 0; i < BT_NUM; i++){
    		for(j = 0; j < num_size[i]; j++){
      			if(C[i][j] != D[i][j]){
        			printf("Error:%f, %f, %d, %d\n", C[i][j], D[i][j], i, j);
				flag = 1;
				break;
      			}
		}
	}

	if(!flag) printf("Verify Successfully\n");

  	for(i = 0; i < BT_NUM; i++){
    		checkCudaErrors(hipHostFree(A[i]));
    		checkCudaErrors(hipFree(A_dev[i]));
   	 	checkCudaErrors(hipHostFree(C[i]));
    		checkCudaErrors(hipFree(C_dev[i]));
    		free(D[i]);
    		checkCudaErrors(hipFree(pos_task_dev[i]));
    		checkCudaErrors(hipFree(d_Stride[i]));

  	}
  	checkCudaErrors(hipFree(num_thread_dev));
  	return 0;
}

__host__ __device__ void SubroutineDCTvector(float *FirstIn, int StepIn, float *FirstOut, int StepOut)
{
    float X07P = FirstIn[0*StepIn] + FirstIn[7*StepIn];
    float X16P = FirstIn[1*StepIn] + FirstIn[6*StepIn];
    float X25P = FirstIn[2*StepIn] + FirstIn[5*StepIn];
    float X34P = FirstIn[3*StepIn] + FirstIn[4*StepIn];

    float X07M = FirstIn[0*StepIn] - FirstIn[7*StepIn];
    float X61M = FirstIn[6*StepIn] - FirstIn[1*StepIn];
    float X25M = FirstIn[2*StepIn] - FirstIn[5*StepIn];
    float X43M = FirstIn[4*StepIn] - FirstIn[3*StepIn];

    float X07P34PP = X07P + X34P;
    float X07P34PM = X07P - X34P;
    float X16P25PP = X16P + X25P;
    float X16P25PM = X16P - X25P;

    FirstOut[0*StepOut] = C_norm * (X07P34PP + X16P25PP);
    FirstOut[2*StepOut] = C_norm * (C_b * X07P34PM + C_e * X16P25PM);
    FirstOut[4*StepOut] = C_norm * (X07P34PP - X16P25PP);
    FirstOut[6*StepOut] = C_norm * (C_e * X07P34PM - C_b * X16P25PM);

    FirstOut[1*StepOut] = C_norm * (C_a * X07M - C_c * X61M + C_d * X25M - C_f * X43M);
    FirstOut[3*StepOut] = C_norm * (C_c * X07M + C_f * X61M - C_a * X25M + C_d * X43M);
    FirstOut[5*StepOut] = C_norm * (C_d * X07M + C_a * X61M + C_f * X25M - C_c * X43M);
    FirstOut[7*StepOut] = C_norm * (C_f * X07M + C_d * X61M + C_c * X25M + C_a * X43M);
}

__host__ __device__ void SubroutineIDCTvector(float *FirstIn, int StepIn, float *FirstOut, int StepOut)
{
    float Y04P   = FirstIn[0*StepIn] + FirstIn[4*StepIn];
    float Y2b6eP = C_b * FirstIn[2*StepIn] + C_e * FirstIn[6*StepIn];

    float Y04P2b6ePP = Y04P + Y2b6eP;
    float Y04P2b6ePM = Y04P - Y2b6eP;
    float Y7f1aP3c5dPP = C_f * FirstIn[7*StepIn] + C_a * FirstIn[1*StepIn] + C_c * FirstIn[3*StepIn] + C_d * FirstIn[5*StepIn];
    float Y7a1fM3d5cMP = C_a * FirstIn[7*StepIn] - C_f * FirstIn[1*StepIn] + C_d * FirstIn[3*StepIn] - C_c * FirstIn[5*StepIn];

    float Y04M   = FirstIn[0*StepIn] - FirstIn[4*StepIn];
    float Y2e6bM = C_e * FirstIn[2*StepIn] - C_b * FirstIn[6*StepIn];

    float Y04M2e6bMP = Y04M + Y2e6bM;
    float Y04M2e6bMM = Y04M - Y2e6bM;
    float Y1c7dM3f5aPM = C_c * FirstIn[1*StepIn] - C_d * FirstIn[7*StepIn] - C_f * FirstIn[3*StepIn] - C_a * FirstIn[5*StepIn];
    float Y1d7cP3a5fMM = C_d * FirstIn[1*StepIn] + C_c * FirstIn[7*StepIn] - C_a * FirstIn[3*StepIn] + C_f * FirstIn[5*StepIn];

    FirstOut[0*StepOut] = C_norm * (Y04P2b6ePP + Y7f1aP3c5dPP);
    FirstOut[7*StepOut] = C_norm * (Y04P2b6ePP - Y7f1aP3c5dPP);
    FirstOut[4*StepOut] = C_norm * (Y04P2b6ePM + Y7a1fM3d5cMP);
    FirstOut[3*StepOut] = C_norm * (Y04P2b6ePM - Y7a1fM3d5cMP);

    FirstOut[1*StepOut] = C_norm * (Y04M2e6bMP + Y1c7dM3f5aPM);
    FirstOut[5*StepOut] = C_norm * (Y04M2e6bMM - Y1d7cP3a5fMM);
    FirstOut[2*StepOut] = C_norm * (Y04M2e6bMM + Y1d7cP3a5fMM);
    FirstOut[6*StepOut] = C_norm * (Y04M2e6bMP - Y1c7dM3f5aPM);
}

void computeDCT(float *fSrc, float *fDst, int *Stride, int *size, int *thread, int index){

    int i, j, t, bk, k;
    int td;
    for(bk = 0; bk < TK_NUM; bk++){
        td = thread[index*TK_NUM+bk];
	for(t = 0; t < td; t++){
        	for(i = 0; i < ((td/BLOCK_SIZE)*(td/BLOCK_SIZE)/td); i++)
        	{
            		//process rows
            		for (k = 0; k < BLOCK_SIZE; k++)
            		{
                		SubroutineDCTvector((float *)fSrc + (((i*td+t)/(td/BLOCK_SIZE))*BLOCK_SIZE+k) * Stride[bk] + (((i*td+t)%(td/BLOCK_SIZE))*BLOCK_SIZE) + size[bk] , 1, fDst + ((((i*td+t)/(td/BLOCK_SIZE))*BLOCK_SIZE)+k) * Stride[bk] + (((i*td+t)%(td/BLOCK_SIZE))*BLOCK_SIZE) + size[bk], 1);
            		}
            		//process columns
            		for (k = 0; k < BLOCK_SIZE; k++)
            		{
                		SubroutineDCTvector(fDst + (((i*td+t)/(td/BLOCK_SIZE)) * BLOCK_SIZE) * Stride[bk] + (((i*td+t)%(td/BLOCK_SIZE))*BLOCK_SIZE+k) + size[bk], Stride[bk], fDst + ((((i*td+t)/(td/BLOCK_SIZE))) * BLOCK_SIZE) * Stride[bk] + (((i*td)%(td/BLOCK_SIZE)) * BLOCK_SIZE + k) + size[bk], Stride[bk]);
            		}
        	}
    	}

    }
}

void computeIDCT(float *fSrc, float *fDst, int *Stride, int *size, int *thread, int index){

    int i, j, t, bk, k;
    int td;
    for(bk = 0; bk < TK_NUM; bk++){
	td = thread[index*TK_NUM+bk];
	for(t = 0; t < td; t++){
          	for(i = 0; i < ((td/BLOCK_SIZE)*(td/BLOCK_SIZE)/td); i++)
          	{
            		//process rows
            		for (k = 0; k < BLOCK_SIZE; k++)
            		{
                		SubroutineIDCTvector((float *)fSrc + (((i*td+t)/(td/BLOCK_SIZE))*BLOCK_SIZE+k) * Stride[bk] + (((i*td+t)%(td/BLOCK_SIZE))*BLOCK_SIZE) + size[bk], 1, fDst + ((((i*td+t)/(td/BLOCK_SIZE))*BLOCK_SIZE)+k) * Stride[bk] + (((i*td+t)%(td/BLOCK_SIZE))*BLOCK_SIZE)+ size[bk], 1);
            		}
            		//process columns
            		for (k = 0; k < BLOCK_SIZE; k++)
            		{
                		SubroutineIDCTvector(fDst + (((i*td+t)/(td/BLOCK_SIZE)) * BLOCK_SIZE) * Stride[bk] + (((i*td+t)%(td/BLOCK_SIZE))*BLOCK_SIZE+k) + size[bk], Stride[bk], fDst + ((((i*td+t)/(td/BLOCK_SIZE))) * BLOCK_SIZE) * Stride[bk] + (((i*td+t)%(td/BLOCK_SIZE)) * BLOCK_SIZE + k) + size[bk], Stride[bk]);
            		}
        	}
    	}

    }
}

__device__ void d_computeIDCT(float *fSrc, float *fDst, int *Stride, int *size, int *thread, int index){

    int t = threadIdx.x;
    int bk = blockIdx.x;
    int i, j, k;
    int td;
    td = thread[index*TK_NUM+bk];
    if(t < td){
	for(i = 0; i < ((td/BLOCK_SIZE)*(td/BLOCK_SIZE)/td); i++)
        {
        	//process rows
                for (k = 0; k < BLOCK_SIZE; k++)
                {
                	SubroutineIDCTvector((float *)fSrc + (((i*td+t)/(td/BLOCK_SIZE))*BLOCK_SIZE+k) * Stride[bk] + (((i*td+t)%(td/BLOCK_SIZE))*BLOCK_SIZE), 1, fDst + ((((i*td+t)/(td/BLOCK_SIZE))*BLOCK_SIZE)+k) * Stride[bk] + (((i*td+t)%(td/BLOCK_SIZE))*BLOCK_SIZE), 1);
                }
                //process columns
                for (k = 0; k < BLOCK_SIZE; k++)
                {
                        SubroutineIDCTvector(fDst + (((i*td+t)/(td/BLOCK_SIZE)) * BLOCK_SIZE) * Stride[bk] + (((i*td+t)%(td/BLOCK_SIZE))*BLOCK_SIZE+k), Stride[bk], fDst + ((((i*td+t)/(td/BLOCK_SIZE))) * BLOCK_SIZE) * Stride[bk] + (((i*td+t)%(td/BLOCK_SIZE)) * BLOCK_SIZE + k), Stride[bk]);
                }
       }

    }
}

__device__ void d_computeDCT(float *fSrc, float *fDst, int *Stride, int *size, int *thread, int index){
    int t = threadIdx.x;
    int bk = blockIdx.x;
    int i, j, k;
    int td;
    td = thread[index*TK_NUM+bk];
    if(t < td){
	for(i = 0; i < ((td/BLOCK_SIZE)*(td/BLOCK_SIZE)/td); i++)
        {
        	//process rows
                for (k = 0; k < BLOCK_SIZE; k++)
                {
                	SubroutineDCTvector((float *)fSrc + (((i*td+t)/(td/BLOCK_SIZE))*BLOCK_SIZE+k) * Stride[bk] + (((i*td+t)%(td/BLOCK_SIZE))*BLOCK_SIZE) + size[bk] , 1, fDst + ((((i*td+t)/(td/BLOCK_SIZE))*BLOCK_SIZE)+k) * Stride[bk] + (((i*td+t)%(td/BLOCK_SIZE))*BLOCK_SIZE) + size[bk], 1);
                }
                //process columns
                for (k = 0; k < BLOCK_SIZE; k++)
                {
                	SubroutineDCTvector(fDst + (((i*td+t)/(td/BLOCK_SIZE)) * BLOCK_SIZE) * Stride[bk] + (((i*td+t)%(td/BLOCK_SIZE))*BLOCK_SIZE+k) + size[bk], Stride[bk], fDst + ((((i*td+t)/(td/BLOCK_SIZE))) * BLOCK_SIZE) * Stride[bk] + (((i*td)%(td/BLOCK_SIZE)) * BLOCK_SIZE + k) + size[bk], Stride[bk]);
                }
        }

    }

}

__global__ void d_DCT(float *fSrc, float *fDst, int *Stride, int *size, int *thread, int index){
    d_computeDCT(fSrc, fDst, Stride, size, thread, index);
    //d_computeIDCT(fSrc, fDst, Stride, size, thread, index);
}

void DCT(float *fSrc, float *fDst, int *Stride, int *size, int *thread, int index){
    computeDCT(fSrc, fDst, Stride, size, thread, index);
    //computeIDCT(fSrc, fDst, Stride, size, thread, index);
}
